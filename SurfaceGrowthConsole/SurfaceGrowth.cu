#include "hip/hip_runtime.h"
//-------------------------------------------------------------
// SurfaceGrowth.cu - contains kernel functions for computing, 
// C wrappers for some of the CUDA API and wrappers for the
// kernels for convenient calling from "SurfaceGrowth.cpp"
// Copyright (c) 2010 Mykola Prodanov
// (this code has been written in Sumy, Ukraine)
//-------------------------------------------------------------
#include "SurfaceGrowth.h"			// includes cuda headers and prototypes

hipDeviceProp_t gDeviceProp;			// device properties

// globals for computing, prefix h - host variable, no prefix or d - device variable
__device__ __constant__ SimParams dparams;		// global device variable in constant memory for GPU
__device__ __constant__ SimParams *pdparams;		// global device variable in constant memory for GPU
__device__	uint count = 0;			// for sum, from Programming guide, p. 111
__shared__	bool isLastBlockDone;	// for sum, from Programming guide, p. 111

__shared__	int K[BLOCK_SIZE];		// BLOCK_SIZE is defined in SurfaceGrowth.h
__shared__	float4 B[BLOCK_SIZE];

//////////////////////////////////////////////////////////
// prototypes of some host functions called from wrappers
//////////////////////////////////////////////////////////

void AccumProps (int icode, SimParams *hparams);	// accumulate properties
void PrintSummary (FILE *fp, SimParams *hparams);	// print results in a file
int CreatePdbFile(char *szFilePath, SimParams *hparams, float4 *r);// create .pdb file
void PrintRdf(SimParams *hparams, uint *hHistRdf);	// prints rdf data in a file
// for random numbers
real RandR (SimParams *hparams);
void VRandRfloat4 (float4 *p, SimParams *hparams);
// host functions for diffusion
void InitDiffusion(TBuf *tBuf, real *rrDiffuseAv, SimParams *hparams);
void ZeroDiffusion(real *rrDiffuseAv, SimParams *hparams);
void PrintDiffusion(real *rrDiffuseAv, FILE *file, SimParams *hparams);
void AccumDiffusion(TBuf *tBuf, real *rrDiffuseAv, FILE *file, SimParams *hparams);
void EvalDiffusion(TBuf *tBuf, real *rrDiffuseAv, FILE *file, SimParams *hparams, float3 centerOfMass);

////////////////////
// device functions
////////////////////
////////////////////////////////////////////////////////////////////////////////////////////
// eam functions (X.W.Zhou, H.N.G.Wadley, R.A.Johnson et al. Acta Materialia 49 (2001) 4005)
////////////////////////////////////////////////////////////////////////////////////////////
// compute phi
__device__ real EamPhi(real r)
{
	real rei = dparams.rei;		// diminish number of calls to constant memory
	real r_rei = r*rei;			// avoid redundant multiplications

	real phi = 
		dparams.A * expf( -dparams.alpha * (r_rei-1.f) ) / 
	( 1.f + powf( (r_rei - dparams.kappa), 20.f) ) -
		dparams.B * expf( -dparams.beta * (r_rei-1.f) ) / 
	( 1.f + powf( (r_rei - dparams.lambda) , 20.f) );

	return phi;
}

// compute f
__device__ real Eamf(real r)
{
	real rei = dparams.rei;
	real r_rei = r*rei;

	return
		dparams.fe*expf( -dparams.beta*(r_rei-1.f) ) / 
		(1.f + powf( (r_rei - dparams.lambda) , 20.f) );
}

// when rho < rhon = 0.85*rhoe
__device__ real EamFrhoSmall(real rho)
{
	//real rhon = 0.85f * dparams.rhoe;
	real rhoni = dparams.rhoei * 1.176471f;	// 1 / (0.85*rhoe)
	real rho_rhoni = rho*rhoni - 1.f;

	return 
		dparams.Fn[0] + 
		dparams.Fn[1]* rho_rhoni + 
		dparams.Fn[2]* rho_rhoni * rho_rhoni +
		dparams.Fn[3]* rho_rhoni * rho_rhoni * rho_rhoni;
}
// when rhon <= rho < rhoo = 1.15*rhoe
__device__ real EamFrhoMedium(real rho)
{	
	real rhoei = dparams.rhoei;
	real rho_rhoei = rho*rhoei - 1.f;

	return 
		dparams.F[0] + 
		dparams.F[1]* rho_rhoei + 
		dparams.F[2]* rho_rhoei * rho_rhoei +
		dparams.F[3]* rho_rhoei * rho_rhoei * rho_rhoei;
}
// when rhoo = 1.15*rhoe <= rho 
__device__ real EamFrhoLarge(real rho)
{
	real rhoei = dparams.rhoei;
	real rho_rhoei = rho*rhoei;

	return
		dparams.Fe * ( 1.f - logf(powf(rho_rhoei, dparams.eta)) ) * powf(rho_rhoei, dparams.eta);
}
// check what embedding functional to use
__device__ real EamF(real rho)
{
	real rhoe = dparams.rhoe;
	real rhon = 0.85f*rhoe;

	if(rho < rhon) return EamFrhoSmall(rho);

	real rhoo = 1.15f*rhoe;
	if( (rho >= rhon) && (rho < rhoo) ) return EamFrhoMedium(rho);

	if( rho >= rhoo ) return EamFrhoLarge(rho);

	return 0.f;
}
////////////////////////////////
// derivatives of eam functions
////////////////////////////////

// dphi / dr
__device__ real EamDPhi(real r)
{
	real rei = dparams.rei;			// diminish number of calls to constant memory
	real alpha = dparams.alpha;
	real beta = dparams.beta;
	real kappa = dparams.kappa;
	real lambda = dparams.lambda;
	real r_rei = r*rei;				// to avoid redundant multiplication

	real denomA = 1.f + powf( (r_rei - kappa), 20.f );
	real denomB = 1.f + powf( (r_rei - lambda) , 20.f );

	return 
		(
		-dparams.A*expf( -alpha*(r_rei-1.f) ) * 
		( alpha + 20.f*powf( (r_rei - kappa),19.f )/denomA ) / denomA +
		dparams.B*expf( -beta*(r_rei-1.f) ) *
		( beta + 20.f*powf( (r_rei - lambda),19.f )/denomB ) / denomB
		)*rei;
}

// df / dr
__device__ real EamDf(real r)
{
	real rei = dparams.rei;
	real beta = dparams.beta;
	real lambda = dparams.lambda;
	real r_rei = r*rei;				// to avoid redundant multiplication

	real denomB = 1.f + powf( (r_rei - lambda) , 20.f);
	
	return
		-dparams.fe * rei * expf(-beta*(r_rei-1.f)) * 
		( beta + 20.f*powf( (r_rei - lambda),19.f )/denomB ) / denomB;
}
// dF / drho when rho < rhon = 0.85*rhoe
__device__ real EamDFrhoSmall(real rho)
{
	real rhoni = dparams.rhoei * 1.176471f;	// 1/rhon = 1 / (0.85*rhoe)
	real rho_rhoni = rho * rhoni - 1.f;			// to avoid redundant multiplication

	return
		rhoni * (
		dparams.Fn[1] + 
		2.f*dparams.Fn[2]* rho_rhoni +
		3.f*dparams.Fn[3]* rho_rhoni * rho_rhoni
		);
}
// dF / drho when rhon <= rho < rhoo = 1.15*rhoe
__device__ real EamDFrhoMedium(real rho)
{
	real rhoei = dparams.rhoei;
	real rho_rhoei = rho*rhoei;

	return
		rhoei * (
		dparams.F[1] + 
		2.f*dparams.F[2]*(rho_rhoei - 1.f) +
		3.f*dparams.F[3]*(rho_rhoei - 1.f)*(rho_rhoei - 1.f)
		);
}
// dF / drho when rhoo = 1.15*rhoe <= rho
__device__ real EamDFrhoLarge(real rho)
{
	real rhoei = dparams.rhoei;
	real eta = dparams.eta;
	real rho_rhoei = rho*rhoei;

	return 
		- dparams.Fe * eta * rhoei * powf(rho_rhoei, (eta-1.f) ) *
		logf(powf(rho_rhoei, eta));
}
// see derivative of what embedding functional to use
__device__ real EamDF(real rho)
{
	real rhoe = dparams.rhoe;
	real rhon = 0.85f*rhoe;

	if(rho < rhon) return EamDFrhoSmall(rho);

	real rhoo = 1.15f*rhoe;
	if( (rho >= rhon) && (rho < rhoo) ) return EamDFrhoMedium(rho);

	if( rho >= rhoo ) return EamDFrhoLarge(rho);

	return 0.f;
}

//////////////////////////////////////////////
// kernels (K at the end of the function name)
//////////////////////////////////////////////

/////////////////////////////////////////
// kernels for generating of coordinates
/////////////////////////////////////////

// initialize coordinates of fcc lattice
// each block is a strip with length of min unit cells, and width and height of 1 cell
// each thread calculates coordinates of atoms in one cell, which contains 4 atoms for fcc
__global__
void InitFccCoordsK(float4 *pos)
{
	float4 gap, c;
	uint j, n, nx, ny, nz;		

	VDiv (gap, dparams.region, dparams.initUcell);		// distance between the cells

	// define atom indexes
	nx = blockIdx.x;
	ny = blockIdx.y;
	nz = threadIdx.z;
	n = 4*(nx + ny * gridDim.x + nz * gridDim.x * gridDim.y);

	// handle threads which do not work
	if(n > dparams.nMol - 4) return;
	VSet (c, nx + 0.25f, ny + 0.25f, nz + 0.25f);	
	
	VMul (c, c, gap);
	VVSAdd (c, -0.5f, dparams.region);
	for (j = 0; j < 4; j ++) {
		pos[n].x = c.x;
		pos[n].y = c.y;
		pos[n].z = c.z;	
		pos[n].w = 0.0f;			
		if (j != 3) {
			if (j != 0) pos[n].x += 0.5f * gap.x;
			if (j != 1) pos[n].y += 0.5f * gap.y;
			if (j != 2) pos[n].z += 0.5f * gap.z;
		}		
		n += 1;
	}	
}

// initialize coordinate of the nanoparticle atoms
// as fcc lattice above the graphene sheet
__global__ void InitSlabCoordsK( float4 *pos )
{
	float4 gap, c;
	uint j, n, nx, ny, nz;
	float3 region;
	real shiftY;
	
	region.x = gridDim.x*dparams.a;
	// number of cells along y * lattice constant
	region.y = gridDim.y*dparams.a;
	// for symmetric coordinates
	if(blockDim.z == 1)	// if one layer	
		shiftY = 0.5f*(region.y - ceilf(0.25f*dparams.nMolMe/gridDim.x) * dparams.a);	
	else	
		shiftY = 0.f;

	region.z = dparams.region.z;// blockDim.z == number of layers of Me

	gap.x = gap.y = gap.z = dparams.a;		// distance between the cells
	
	// define atom indexes
	nx = blockIdx.x;
	ny = blockIdx.y;
	nz = threadIdx.z;
	n = 4*(nx + ny * gridDim.x + nz * gridDim.x * gridDim.y);
	
	// handle threads which do not work
	VSet (c, nx + 0.25f, ny + 0.25f, nz + 0.25f);		
	VMul (c, c, gap);
	VVSAdd (c, -0.5f, region);	
	for (j = 0; j < 4; j ++) {	
		if(n < dparams.nMolMe) 
		{
			pos[n].x = c.x;
			pos[n].y = c.y + shiftY;
			pos[n].z = c.z + 0.5f*dparams.a + dparams.cellShiftZ/dparams.invWidth.z;	
			pos[n].w = 0.0f;			
			if (j != 3) {
				if (j != 0) pos[n].x += 0.5f * gap.x;
				if (j != 1) pos[n].y += 0.5f * gap.y;
				if (j != 2) pos[n].z += 0.5f * gap.z;
			}			
			n += 1;
		}	
	}
}

__global__
void InitGrapheneCoordsK(float4 *pos)
{    
//  6	    7		// this is the numeration of atoms (rotated by 90 anticlockwise)
//	\	   /
//	 \4___/5
//	 /    \  
// 3/      \0
//	\	   /
//	2\____/1		// axes: y from right to left, x from bottom to top in this figure

	real dx, dy, x, y;
	int	n, cx, cy;

	real cos30 = 0.8660254038f;

	// find index of the atom
	n = blockIdx.x*blockDim.x + threadIdx.x;	// blockDim.x is assumed == 32

	if( n < (dparams.nMol-dparams.nMolMe) )
	{
		// find 2D index of the block in the grid of blocks
		cy = blockIdx.x / dparams.initUcell.x;
		cx = blockIdx.x - cy * dparams.initUcell.x;

		// all coordinates in the block are shifted by the following quantities
		dx = cx*8.f*cos30;	// here should be * a = 1.42 angstom, but it is = 1 in our units
		dy = cy*6.f;

		// find index of the thread in a 2D grid of blockDim.x threads
		cy = threadIdx.x / 8.f;	// each cell handles 8 atoms, there are blockDim.x/8 cells
		cx = threadIdx.x - cy*8.f;

		// see what atom we have
		switch(cx)	{
	case 0:
		x = cos30;
		y = 0.f;
		break;
	case 1:
		x = 0.f;
		y = 0.5f;
		break;
	case 2:
		x = 0.f;
		y = 1.5f;
		break;
	case 3:
		x = cos30;
		y = 2.f;
		break;
	case 4:
		x = 2.f*cos30;
		y = 1.5f;
		break;
	case 5:
		x = 2.f*cos30;
		y = 0.5f;
		break;
	case 6:
		x = 3.f*cos30;
		y = 2.f;
		break;
	case 7:
		x = 3.f*cos30;
		y = 0.f;
		break;
		}

		// additional shift dependently on their grid number due to thread
		if( cy == 1 )
			dx += 4.f*cos30;
		else if( cy == 2 )
			dy += 3.f;
		else if( cy == 3 ) {
		dx += 4.f*cos30;
		dy += 3.f;
		}		

		// shift the origin to the center of the layer
		dx = dx + 0.5f*(cos30 - dparams.region.x);
		dy = dy + 0.5f*(1.f - dparams.region.y);

		// save coordinates
		pos[n + dparams.nMolMe].x = x + dx;			// !! note shift in indexes due to metal
		pos[n + dparams.nMolMe].y = y + dy;
		pos[n + dparams.nMolMe].z = dparams.z_0;	
		pos[n + dparams.nMolMe].w = 0.0f;			// zero energy		
	}
}

// integrate equations of motion using Verlet method
__global__ void LeapfrogStepK (int part, float4 *dr, float3 *dv, float3 *da)
{
	// index of molecule for current thread
	int n = blockIdx.x * blockDim.x + threadIdx.x;
	
	if( n < dparams.nMolMe )	// for metal process all atoms
	{
		if (part == 1)	{
			VVSAdd (dv[n], 0.5f * dparams.deltaT, da[n]);
			VVSAdd (dr[n], dparams.deltaT, dv[n]);    
		} 
		else {	
			VVSAdd (dv[n], 0.5f * dparams.deltaT, da[n]);
		}
	}
	else if( (n >= dparams.nMolMe ) && 
		( n < dparams.nMol ) )// for graphene don't process boundary atoms
	{
		real xLeft, yBottom, xRight, yTop;
		real cos30 = cos(M_PI / 6.f);

		xLeft = 0.5f*(cos30 - dparams.region.x);
		xRight = 0.5f*(dparams.region.x - cos30);
		yBottom = 0.5f*(1.f - dparams.region.y);
		yTop = 0.5f*(dparams.region.y - 1.f);

		if( (dr[n].x != xLeft) && (dr[n].x < xRight) && 
			(dr[n].y != yTop) && (dr[n].y != yBottom) )
		{
			if (part == 1) {
				VVSAdd (dv[n], 0.5f * dparams.deltaT, da[n]);
				VVSAdd (dr[n], dparams.deltaT, dv[n]);    
			} else {
				VVSAdd (dv[n], 0.5f * dparams.deltaT, da[n]);
			}
		}
	}	// end else if( (n >= dparams.nMolMe ) 
}

__global__ void ApplyBoundaryCondK( float4 *dr )
{
	// index of molecule for current thread
	int n = blockIdx.x * blockDim.x + threadIdx.x;
	// don't perform redundant threads to prevent memory overwriting
	if(n < dparams.nMol) {
		// type manually without macroses to avoid problems
		// x
		if (dr[n].x >= 0.5f * dparams.region.x)      dr[n].x -= dparams.region.x;
		else if (dr[n].x < -0.5f * dparams.region.x) dr[n].x += dparams.region.x;
		// y
		if (dr[n].y >= 0.5f * dparams.region.y)      dr[n].y -= dparams.region.y;
		else if (dr[n].y < -0.5f * dparams.region.y) dr[n].y += dparams.region.y;
		// z
		if (dr[n].z >= 0.5f * dparams.region.z)      dr[n].z -= dparams.region.z;
		else if (dr[n].z < -0.5f * dparams.region.z) dr[n].z += dparams.region.z;
	}
}

__global__ void ApplyBerendsenThermostat( float3 *dv, real *vvSum, int stepCount )
{
	// index of molecule for current thread
	int n = blockIdx.x * blockDim.x + threadIdx.x;
	real beta;

	// !consider only carbon atoms
	if( (n >= dparams.nMolMe) && (n < dparams.nMol) )
	{
		real kinEnergy = (*vvSum)*0.5f / dparams.nMol ;
		beta = 
			sqrtf( 1.f + dparams.gammaBerendsen *
			(dparams.temperature*1.5f*dparams.kB/kinEnergy - 1.f) );
		dv[n].x = beta * dv[n].x;
		dv[n].y = beta * dv[n].y;
		dv[n].z = beta * dv[n].z;
	}
	// for metal atoms
	if( (dparams.iRegime == 2)&&(n < dparams.nMolMe)&&
		(stepCount > dparams.stepEquil)&&		// this is redundant
		(stepCount < dparams.stepEquil + dparams.stepCool) )
	{
		real kinEnergy = (*vvSum)*0.5f / dparams.nMol ;
		beta = 
			sqrtf( 1.f + dparams.gammaBerendsen *
			(dparams.temperature*dparams.kB/kinEnergy - 1.f) );
		dv[n].x = beta * dv[n].x;
		dv[n].y = beta * dv[n].y;
		dv[n].z = beta * dv[n].z;
	}	
}

// deposit atoms (for Surface Growth regime)
__global__ void InsertAtomsK( float4 *dr, float3 *dv, int nMolDeposited, int nMolToDeposit )
{
	real z = 0.49f * dparams.region.z;

	// index of molecule for current thread
	int n = blockIdx.x * blockDim.x + threadIdx.x;

	int iDeposited = nMolDeposited;
	int iInterval = iDeposited + nMolToDeposit;	

	if(n < dparams.nMolMe)		// process only metalic atoms
	{
		if( (n >= iDeposited) && (n < iInterval ) )	//atoms to deposit
		{			
			dr[n].z = z;
			// give initial velocity
			dv[n].z = -dparams.velMagDepos;// !note minus sign
		}
		if(n >= iInterval)		// it is early to deposit these atoms
		{			
			dr[n].z = 1.5f*dparams.region.z + 0.5f*(n+1)*dparams.region.z;
			dv[n].x = 0.f;
			dv[n].y = 0.f;
			dv[n].z = 0.f;
		}
	}
}

// apply shear (for Shear regime)
__global__ void ApplyShearK( float4 *dr, float3 *da, real shear, real centerOfMassX, 
							uint *numOfSharedMols )
{
	// index of molecule for current thread
	int n = blockIdx.x * blockDim.x + threadIdx.x;
	
	if( n < dparams.nMolMe)
	{
		real deltaR = centerOfMassX - dr[n].x;
		// apply shear only to atoms that are to the left to the center of mass
		if( deltaR > 0.f)
		{
			da[n].x += shear;
			unsigned int val = 1;
			atomicAdd((unsigned int *)numOfSharedMols, val);
		}
	}	
}

//////////////////////////////////////////////
// kernels assosiated with force calculations
//////////////////////////////////////////////

// build cells
// each block finds particles that belong to the cell with the number equal to blockIdx.x;
// this is done as follows: for each step a thread reads particle coordinates, 
// so each such a step the block reads blockDim.x particles. Then each thread calculates
// index of the cell of the particle, and compares it with the blockIdx.x. If it is =,
// than the thread finds free place in CELL array by atomically enlarging number of
// atoms in the cell, and then stores atomically index at that place
__global__ void BinAtomsIntoCellsK(float4 *dr, int *CELL, uint *molsInCells)
{	
	float4	rShifted, r;					// shifted and initial coordinates
	int3	cc;								// 3D index of the cell
	int		c, whereToWrite, n;				// counters	

	int		count = 0;						// reads have been made
	int		numOfIter = floor( (float)dparams.nMol/blockDim.x );	// total number of reads

	while( 1 )
	{
		n = blockDim.x*count + threadIdx.x;	// get particle index for current thread				
		
		//check whether this is the last cycle
		if( count == numOfIter )
		{
			if( n < dparams.nMol )	r = dr[n];	// process partially filled portions
			else VZero(r);						// virtual particle, fill with 0

			// find index of the cell
			// shift coordinates into 1st quadrant
			rShifted.x = r.x + 0.5f * dparams.region.x;
			rShifted.y = r.y + 0.5f * dparams.region.y;
			rShifted.z = r.z + 0.5f * dparams.region.z;

			// find index of the cell in 3D x, y, z reprezentation
			VMul (cc, rShifted, dparams.invWidth);	// invWidth is placed in dparams to avoid division each time
			c = VLinear (cc, dparams.cells);		// convert index of the cell into 1D			
			
			if( n < dparams.nMol )	// if the particle is not virtual
			{
				if( c == blockIdx.x )	// if the particle is in the desired cell
				{
					// !note here atomicAdd, but not atomicInc !
					// it returns the previous nonincremented value, which is what we need
					whereToWrite = atomicAdd(&molsInCells[c], 1);	// find shift in the array
					CELL[ c * blockDim.x + whereToWrite ] = n;				
				}
			}	// end if( n < dparams.nMol )

			break;	// terminate loop while
		}	// end if( count == numOfIter )

		// if count < numOfIter
		r = dr[n];
		// find index of the cell
		// shift coordinates into 1st quadrant
		rShifted.x = r.x + 0.5f * dparams.region.x;
		rShifted.y = r.y + 0.5f * dparams.region.y;
		rShifted.z = r.z + 0.5f * dparams.region.z;

		// find index of the cell in 3D x, y, z reprezentation
		VMul (cc, rShifted, dparams.invWidth);	// invWidth is placed in params to avoid division each time
		c = VLinear (cc, dparams.cells);		// convert index of the cell into 1D

		if( c == blockIdx.x )	// if the particel is in the desired cell
		{
			whereToWrite = atomicAdd(&molsInCells[c], 1);	// find shift in the array			
			CELL[ c * blockDim.x + whereToWrite ] = n;		// save particle index			
		}
		++count;			
	}	// end while( 1 )	
}

// build neighbor list, algorithm 4 from Anderson et al.
__global__ void BuildNebrListK (float4 *dr,		// coordinates of molecules (global)							  
							   int *CELL,		// indexes of molecules (global)
							   int *NN,			// number of neighbors (global)
							   int *NBL)		// neighbor list (global)							  
{		
	int3 cc, m2v, vOff[] = OFFSET_VALS_LONG;	// 3D indexes of the cells
	int i, j, offset, n;						// counters
	int C;										// 1D cell indexes
	int nNeigh;									// number of neighbors, nebrTabLength in Rapaport
	float3 shift;								// variable for handling boundary conditions
	float4 A, deltaR;							// radus-vectors

	// define 3D index of the current cell, blockIdx.x is 1D index of the cell
	cc.z = blockIdx.x / (dparams.cells.x * dparams.cells.y);
	cc.y = (blockIdx.x - cc.z * dparams.cells.x * dparams.cells.y) / dparams.cells.x;
	cc.x = blockIdx.x - (cc.z*dparams.cells.y + cc.y)*dparams.cells.x;
	
// here begins implementation of the Algorithm 4 from Anderson
	n = blockIdx.x * blockDim.x + threadIdx.x;
	if(n < dparams.maxMol)
		i = CELL[blockIdx.x * blockDim.x + threadIdx.x];// step 1, get particle index
	else i = -1;
	nNeigh = 0;										// step 2
	if(i != -1)	{									// added by me to avoid memory problems
		A = dr[i];									// step 3			
	}
	// begin step 4
	for (offset = 0; offset < N_OFFSET_LONG; offset ++)	// loop over all 27 neighboring cells 
	{
		VAdd (m2v, cc, vOff[offset]);			// find 3D index of the neighboring cell
		
		shift.x = 0.f;							// zero shift for boundary conditions
		shift.y = 0.f;
		shift.z = 0.f;

		// apply boundary conditions
		// !type manually instead of using macroses, because they may not work!
		// x
		if (m2v.x >= dparams.cells.x) { 
			m2v.x = 0; 
			shift.x = dparams.region.x; 
		} else if (m2v.x < 0) {
			m2v.x = dparams.cells.x - 1;
			shift.x = - dparams.region.x;
		}
		// y
		if (m2v.y >= dparams.cells.y) { 
			m2v.y = 0; 
			shift.y = dparams.region.y; 
		} else if (m2v.y < 0) {
			m2v.y = dparams.cells.y - 1;
			shift.y = - dparams.region.y;
		}
		// z
		if (m2v.z >= dparams.cells.z) { 
			m2v.z = 0; 
			shift.z = dparams.region.z; 
		} else if (m2v.z < 0) {
			m2v.z = dparams.cells.z - 1;
			shift.z = - dparams.region.z;
		}

		C = VLinear (m2v, dparams.cells);			// find 1D index of the neighboring cell
	// end step 4

		__syncthreads();							// step 5
		K[threadIdx.x] = CELL[C * blockDim.x + threadIdx.x];	// step 6
		if(K[threadIdx.x] != -1)					// added by me, to avoid memory error
			B[threadIdx.x] = dr[ K[threadIdx.x] ];	// step 7
		else
			VZero(B[threadIdx.x]);					// added by me
		__syncthreads();							// step 8		
		
		if( i != -1)	{							// step 9
			for(j = 0; j < blockDim.x; j++)	
			{
				// step 10, loop over atoms from the current neighboring cell C
				if( K[j] != -1 )	{				// steps 11 - 13,					
					deltaR.x = A.x - B[j].x;		// step 14
					deltaR.y = A.y - B[j].y;
					deltaR.z = A.z - B[j].z;
					
					deltaR.x = deltaR.x - shift.x;	// step 15, boundary conditions
					deltaR.y = deltaR.y - shift.y;
					deltaR.z = deltaR.z - shift.z;
					
					if ( ((deltaR.x*deltaR.x + deltaR.y*deltaR.y + deltaR.z*deltaR.z) < 
						dparams.rrNebr) && (i != K[j]) )	// step 16
					{						
						NBL[nNeigh * dparams.nMol + i] = K[j];	// step 17
						++nNeigh;								// step 18
					}											// step 19
				}	// end if( K[j] != -1 )
			}													// step 20 
		}														// step 21	
	} // step 22 end for (offset = 0; offset < N_OFFSET_LARGE; offset ++) 		

	if(i != -1)
		NN[i] = nNeigh;		// step 23	
}

// compute rho for eam potential
// algorithm is the same as Compute Forces in Anderson;
// the use of separate kernel for this is to avoid thread syncronization
__global__ void EamComputeRhoK(	real	*rho,			// electron density (out)								  
								float4	*dr,			// array of coordinates (global)
								int		*NN,			// array of number of neighbors
								int		*NBL)			// neigbors list	
{	
	float4	A, B, deltaR;					// coordinates
	real	rr, rhoVal, rhoSum;				// scalars
	int		nNeigh;							// number of neighbors
	int		j, k;							// counters

	// zero quntities
	rhoSum = 0.f;	
	rhoVal = 0.f;

// begin Algorithm 2 from Anderson
	// index of molecule for current thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;	// step 1
	// do not perform redundant threads
	// !here nMolMe
	if(i < dparams.nMolMe) 
	{		
		A = dr[i];						// step 3, load coordinates of atom i
		nNeigh = NN[i];					// step 4, get the number of neighbors for atom i

		for(j = 0; j < nNeigh; j++)		// step 5, loop over all neighbors
		{
			k = NBL[j * dparams.nMol + i];	// step 6
			B = dr[k];						// step 7

			// begin step 8		
			deltaR.x = A.x - B.x;
			deltaR.y = A.y - B.y;
			deltaR.z = A.z - B.z;

			// periodic boundaries
			// x
			if (deltaR.x >= 0.5f * dparams.region.x)      deltaR.x -= dparams.region.x; 
			else if (deltaR.x < -0.5f * dparams.region.x) deltaR.x += dparams.region.x;
			// y
			if (deltaR.y >= 0.5f * dparams.region.y)      deltaR.y -= dparams.region.y; 
			else if (deltaR.y < -0.5f * dparams.region.y) deltaR.y += dparams.region.y;
			// z
			if (deltaR.z >= 0.5f * dparams.region.z)      deltaR.z -= dparams.region.z; 
			else if (deltaR.z < -0.5f * dparams.region.z) deltaR.z += dparams.region.z;
			// end step 8

			// begin step 9
			rr = deltaR.x*deltaR.x + deltaR.y*deltaR.y + deltaR.z*deltaR.z;
			if ( (rr < dparams.rrCutEam) && (k < dparams.nMolMe) )
			{
				rhoVal = Eamf( sqrt(rr) );
				rhoSum += rhoVal;
			}			
		}		// step 14
		rho[i] = rhoSum;		
// end Algorithm 2
	}	// end if(i < dparams.nMolMe)
}

// compute all forces based on the Algorithm 2 from Anderson et al.
__global__ void ComputeForcesK (	float3	*a,				// acceleration (out)								
									float4	*dr,			// array of coordinates (global)
									int	*NN,				// array of number of neighbors
									int	*NBL,				// neigbors list	
									real *rho,				// electron density for metal atom
									real *fForce )			// friction force (out)
{
	float3	fSum, C;			// forces
	float4	A, B, deltaR;		// coordinates
	real	rr, rri, rri3, fcVal, 
			r, rhoi, rhok;		// scalars
	real	uSum, uVal;			// potential energy
	int		nNeigh;				// number of neighbors
	int		j, k;				// counters

	float4	rnn[3];				// coordinates of the nearest neighbors of the current atom
	real	absRnn[3],			// modules of dist
			deltaZ[3], dZ,		// changes in applicate
			rjirki,				// scalar product divided by product of modules
			theta,				// angle between bonds
			tmp, dTheta;
	int		counti = 0;			// number of nearest neighbors

	int		nnk;				// neighbors of the nearest neighbor
	int		countk, m, l;		// counts
	float4	rm;					// coordinates of the second neighbor
	float3	frictForceVal;

	int		atomType;			// if == 0 then metal, if == 1, then carbon

	// zero quntities
	uSum = 0.f;	

	// index of molecule for current thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;	// step 1
	// define what atom we have
	if( i < dparams.nMolMe ) atomType = 0;	// metal
	if( (i >= dparams.nMolMe) && (i < dparams.nMol) ) atomType = 1; // carbon

	// do not perform redundant threads
	if(i < dparams.nMol) 
	{
		a[i].x = 0.f;
		a[i].y = 0.f;
		a[i].z = 0.f;

		// zero friction force
		VZero(frictForceVal);

		VZero(fSum);					// step 2
		A = dr[i];						// step 3
		nNeigh = NN[i];					// step 4
		if( atomType == 0 )
			rhoi = rho[i];					// load electron density if metal atom

		for(j = 0; j < nNeigh; j++)		// step 5
		{
			k = NBL[j * dparams.nMol + i];	// step 6
			B = dr[k];						// step 7
			
			deltaR.x = A.x - B.x;
			deltaR.y = A.y - B.y;
			deltaR.z = A.z - B.z;

			// periodic boundaries
			// x
			if (deltaR.x >= 0.5f * dparams.region.x)      deltaR.x -= dparams.region.x; 
			else if (deltaR.x < -0.5f * dparams.region.x) deltaR.x += dparams.region.x;
			// y
			if (deltaR.y >= 0.5f * dparams.region.y)      deltaR.y -= dparams.region.y; 
			else if (deltaR.y < -0.5f * dparams.region.y) deltaR.y += dparams.region.y;
			// z
			if (deltaR.z >= 0.5f * dparams.region.z)      deltaR.z -= dparams.region.z; 
			else if (deltaR.z < -0.5f * dparams.region.z) deltaR.z += dparams.region.z;
			// square of distance between atoms i and k
			rr = deltaR.x*deltaR.x + deltaR.y*deltaR.y + deltaR.z*deltaR.z;

			// compute force for the current atomic type
			switch( atomType )
			{
				case 0:							// metal
					if( k < dparams.nMolMe )	// neighbor is also metal, so compute EAM
					{
						r = sqrt( rr );
						rhok = rho[k];			// load electron density for atom k from global memory
						// compute paiwise interaction and energy
						fcVal = -EamDPhi( r );
						uVal = EamPhi( r );
						// compute interactions due to the embedding contribution
						fcVal = fcVal - EamDf( r ) *( EamDF(rhoi) + EamDF(rhok) );
						// very important! divide by module of the interatomic distance!
						// to normalize radius vector!
						fcVal = fcVal / r;			

						if (rr >= dparams.rrCutEam) fcVal = 0.0f;	// steps 10 - 12

						VSCopy (C, fcVal, deltaR);		// make vector
						VVAdd(fSum, C);		// step 13

						// added by me: compute energy
						if (rr < dparams.rrCutEam) uSum += uVal;
					}
					else						// neighbor is carbon, so compute LJ
					{
						rri = dparams.sigmaLJ*dparams.sigmaLJ / rr;		
						rri3 = Cube (rri);
						fcVal = 48.f * dparams.epsLJ * rri3 * (rri3 - 0.5f) * rri 
								/ (dparams.sigmaLJ*dparams.sigmaLJ);		
						uVal = 4.f * dparams.epsLJ * rri3 * (rri3 - 1.f);		
						// end step 9

						if (rr >= dparams.rrCutLJ) fcVal = 0.0f;	// steps 10 - 12
						// added by me compute energy
						if (rr < dparams.rrCutLJ) uSum += uVal;

						VSCopy (C, fcVal, deltaR);	

						VVAdd(fSum, C);		// step 13
						// save also friction force
						VVAdd(frictForceVal, C);
					}
					break;	// end metal

				case 1:							// carbon
					if( k < dparams.nMolMe )	// neighbor is metal, so compute LJ
					{
						rri = dparams.sigmaLJ*dparams.sigmaLJ / rr;		
						rri3 = Cube (rri);
						fcVal = 48.f * dparams.epsLJ * rri3 * (rri3 - 0.5f) * rri 
								/ (dparams.sigmaLJ*dparams.sigmaLJ);		
						uVal = 4.f * dparams.epsLJ * rri3 * (rri3 - 1.f);		
						// end step 9

						if (rr >= dparams.rrCutLJ) fcVal = 0.0f;	// steps 10 - 12
						// added by me compute energy
						if (rr < dparams.rrCutLJ) uSum += uVal;

						VSCopy (C, fcVal, deltaR);	

						VVAdd(fSum, C);		// step 13
						
					}
					else				// neighbor is also carbon, so compute spring force
					{
						// see whether this is the nearest neighbor
						if ( (rr < dparams.rrCutC) && (counti < 3) )	
						{
							absRnn[counti] = sqrt(rr);
							deltaZ[counti] = B.z - dparams.z_0;
							rnn[counti].x = deltaR.x;	
							rnn[counti].y = deltaR.y;	
							rnn[counti].z = deltaR.z;				
							
							// get number of neighbors for atom k
							nnk = NN[k];	
							countk = 0;		// number of nearest neighbors of atom k
							for(l = 0; l < nnk; l++)	// loop over neighbors of atom k
							{
								m = NBL[l * dparams.nMol + k];
								rm = dr[m];			// get neighbor's coordinate
								// note that for periodic boundaries we use rk - rm
								// but in forces we will use rm - rk and also use ri - rk
								deltaR.x = B.x - rm.x;
								deltaR.y = B.y - rm.y;
								deltaR.z = B.z - rm.z;
								// periodic boundaries
								// x
								if (deltaR.x >= 0.5f * dparams.region.x)      deltaR.x -= dparams.region.x; 
								else if (deltaR.x < -0.5f * dparams.region.x) deltaR.x += dparams.region.x;
								// y
								if (deltaR.y >= 0.5f * dparams.region.y)      deltaR.y -= dparams.region.y; 
								else if (deltaR.y < -0.5f * dparams.region.y) deltaR.y += dparams.region.y;
								// z
								if (deltaR.z >= 0.5f * dparams.region.z)      deltaR.z -= dparams.region.z; 
								else if (deltaR.z < -0.5f * dparams.region.z) deltaR.z += dparams.region.z;
								
								rr = deltaR.x*deltaR.x + deltaR.y*deltaR.y + deltaR.z*deltaR.z;
								if ((rr < dparams.rrCutC)&&(m != i))	// see whether this is the nearest neighbor
								{
									// compute angular part of the force for the neigbor atom
									VScale(deltaR, (-1.f));	// inverse direction of distance
									rjirki = VDot(rnn[counti],deltaR)/(absRnn[counti] * rr);
									theta = acosf(rjirki);
									dTheta = theta - dparams.theta_0;
									// note that there should be r_0*r_0, but in our units it is = 1.
									tmp = dparams.mu_theta * dTheta / sqrtf(1.f-rjirki*rjirki);
									fcVal = tmp/(absRnn[counti] * rr);
									VSCopy (C, fcVal, deltaR);	
									VVAdd(fSum, C);
									fcVal = -tmp*rjirki/(absRnn[counti] * absRnn[counti]);
									VSCopy (C, (fcVal), rnn[counti]);	
									VVAdd(fSum, C);
									// !potential energy is not computed!

									// z contribution						
									fcVal =	-dparams.mu_p*(rm.z - dparams.z_0)/9.f;
									C.x = 0.f;
									C.y = 0.f;
									C.z = fcVal;		
									VVAdd(fSum, C);									

									++ countk;	// enlarge number of nearest neighbors of atom k
								}
								// use goto, because we want terminate only one loop, but break will terminate all loops
								if( countk == 2 ) goto label;	// exit if 2 nn (except i) have been processed								
							} // end loop over neighbors of atom k
label:
							++ counti;	// enlarge number of nearest neighbors of atom i
						}	// end if (rr < dparams.rrCutC) && (counti < 3)
						if (counti == 3) 
						{
							// compute quantities associated with nearest neighbors
							// compute radial part
							for(j = 0; j < counti; j++)
							{
								// note that we use 1, because r_0 = 1.42 angstrom = 1 dimensionless
								fcVal = -dparams.mu_r*(absRnn[j] - 1.f)/absRnn[j];
								VSCopy (C, (fcVal), rnn[j]);	
								VVAdd(fSum, C);
								// potential energy
								uVal = dparams.mu_r*(absRnn[j] - 1.f)*(absRnn[j] - 1.f);
								uSum += uVal;
							}

							// compute contribution assosiated with z coordinate
							dZ = 2.f*(A.z-dparams.z_0)-(deltaZ[0] + deltaZ[1] + deltaZ[2]);
							fcVal =	-0.666666667f*dparams.mu_p*dZ;
							C.x = 0.f;
							C.y = 0.f;
							C.z = fcVal;		
							VVAdd(fSum, C);
							// potential energy
							dZ = (A.z-dparams.z_0)-0.33333333f*(deltaZ[0] + deltaZ[1] + deltaZ[2]);
							uVal = dparams.mu_p*dZ*dZ;
							uSum += uVal;		

							// compute contribution associated with bending bonds and angle	theta
							// vectors 0 and 1							
							rjirki = VDot(rnn[0],rnn[1])/(absRnn[0] * absRnn[1]);
							theta = acosf(rjirki);
							dTheta = theta - dparams.theta_0;
							// note that there should be r_0*r_0, but in our units it is = 1.
							tmp = dparams.mu_theta * dTheta / sqrtf(1.f-rjirki*rjirki);
							fcVal = tmp*(1.f - absRnn[0]*rjirki/absRnn[1])/(absRnn[0]*absRnn[1]);
							VSCopy (C, (fcVal), rnn[1]);	
							VVAdd(fSum, C);
							fcVal = tmp*(1.f - absRnn[1]*rjirki/absRnn[0])/(absRnn[0]*absRnn[1]);
							VSCopy (C, (fcVal), rnn[0]);	// !here rnn[0] 
							VVAdd(fSum, C);
							// potential energy
							uVal = dparams.mu_theta * dTheta * dTheta;
							uSum += uVal;

							// vectors 1 and 2
							rjirki = VDot(rnn[1],rnn[2])/(absRnn[1] * absRnn[2]);
							theta = acosf(rjirki);
							dTheta = theta - dparams.theta_0;
							// note that there should be r_0*r_0, but in our units it is = 1.
							tmp = dparams.mu_theta * dTheta / sqrtf(1.f-rjirki*rjirki);
							fcVal = tmp*(1.f - absRnn[1]*rjirki/absRnn[2])/(absRnn[1]*absRnn[2]);
							VSCopy (C, (fcVal), rnn[2]);	
							VVAdd(fSum, C);
							fcVal = tmp*(1 - absRnn[2]*rjirki/absRnn[1])/(absRnn[1]*absRnn[2]);
							VSCopy (C, (fcVal), rnn[1]);	
							VVAdd(fSum, C);
							// potential energy
							uVal = dparams.mu_theta * dTheta * dTheta;
							uSum += uVal;

							// vectors 2 and 0
							rjirki = VDot(rnn[2],rnn[0])/(absRnn[2] * absRnn[0]);
							theta = acosf(rjirki);
							dTheta = theta - dparams.theta_0;
							// note that there should be r_0*r_0, but in our units it is = 1.
							tmp = dparams.mu_theta * dTheta / sqrtf(1.f-rjirki*rjirki);
							fcVal = tmp*(1.f - absRnn[2]*rjirki/absRnn[0])/(absRnn[2]*absRnn[0]);
							VSCopy (C, (fcVal), rnn[0]);	
							VVAdd(fSum, C);
							fcVal = tmp*(1.f - absRnn[0]*rjirki/absRnn[2])/(absRnn[0]*absRnn[2]);
							VSCopy (C, (fcVal), rnn[2]);	
							VVAdd(fSum, C);
							// potential energy
							uVal = dparams.mu_theta * dTheta * dTheta;
							uSum += uVal;
							++ counti;	// to prevent following computations
						}	// end if (counti == 3)
					}	// end else		// neighbor is also carbon, so spring force
					break;	// end carbon

			}	// end switch( atomType )
		
		}	// end for(j = 0; j < nNeigh; j++)
		
		if( atomType == 0 )
		{
			a[i].x = fSum.x / dparams.massMe;			//! divide by metallic mass
			a[i].y = fSum.y / dparams.massMe;
			a[i].z = fSum.z / dparams.massMe;
			// save potential energy in .w component of coordinate
			// note: we use 0.5 for pairwise energy 
			// but don't use 0.5 for embedded energy in contrast to Rapaport
			// so don't use 0.5 in EvalProps!
			dr[i].w = 0.5f*uSum + EamF(rhoi);	
			// !save friction force only if the correct regime and metal atoms!
			if(fForce != 0)
				fForce[i] = frictForceVal.x;			
		}
		else if( atomType == 1 )
		{
			a[i] = fSum;
			// save potential energy in .w component of coordinate
			dr[i].w = 0.5f*uSum;			// !don't forget about 0.5 			
		}
// end Algorithm 2
	}	// end if(i < dparams.nMol)
}

////////////////////////////////////////////////////////////////////////////////////
// kernels for evaluation of properties (impulse, kinetic energy, etc.) - reduction
////////////////////////////////////////////////////////////////////////////////////

// compute total impulse
// each block first sums a subset of the array and stores the result in global memory;
// when all blocks are done, the last block done reads each of these partial sums 
// and sums them
__global__ void ComputeVSumK( float3 *dv,	// array of velocities
						float3 *hlpArray )	// helper array, size of ceil(nMol / (2*blockDim.x)) is assumed
{
	// we assume that each block at first computes partial sum of 2*blockDim.x elements
	extern __shared__ float3 partialSum[];	// size of 2*blockDim.x is assumed	
	uint stride, t = threadIdx.x;
	uint maxThreadIdx;	

	int maxBlockIdx = floor(dparams.nMol*0.5f/blockDim.x);

	// for partially filled block define the index of the last thread
	if(maxBlockIdx == 0) maxThreadIdx = dparams.nMol;
	else
		maxThreadIdx = (uint) dparams.nMol % (2*blockDim.x);

	// copy data from global memory to shared
	partialSum[t] = dv[2*blockIdx.x*blockDim.x + t];
	partialSum[blockDim.x+t] = dv[(2*blockIdx.x + 1)*blockDim.x + t];

	// if metal scale by mass	
	float m = dparams.massMe;
	// here not threadIdx, but index of molecule!
	if( (2*blockIdx.x*blockDim.x + t) < dparams.nMolMe )
		partialSum[t].x = m*partialSum[t].x;	
	if( ( (2*blockIdx.x+1)*blockDim.x + t) < dparams.nMolMe )
		partialSum[blockDim.x+t].x = m*partialSum[blockDim.x+t].x;

	// check if the thread is above the range then zero the sum
	// this causes divergent warp
	if( (blockIdx.x == maxBlockIdx) && (t >= maxThreadIdx ) ) 
		VZero(partialSum[t]);
	if( (blockIdx.x == maxBlockIdx) && ((blockDim.x+t) >= maxThreadIdx) ) 
		VZero(partialSum[blockDim.x+t]);

	// check if atom is above the range, then zero sums
	if( (2*blockIdx.x*blockDim.x + t) >= dparams.nMol){
		VZero(partialSum[t]);
	}
	if( ((2*blockIdx.x + 1)*blockDim.x + t) >= dparams.nMol){		
		VZero(partialSum[blockDim.x+t]);
	}

	// begin summation, algorithm from lecture 13 Urbana, Illinois
	// !there were two bugs in the lecture: 
	// 1)should be stride >= 1, but not stride > 1;
	// 2) stride = stride >> 1, but not stride >> 1
	for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
		__syncthreads();
		if( t < stride ) VVAdd(partialSum[t], partialSum[t + stride]);
	}

	// in each block thread t == 0 contains partial sum
	// copy it to global memory (see p. 111 of programming guide)
	if( t == 0)	{
		hlpArray[blockIdx.x] = partialSum[t];		
		__threadfence();	// ensure that the result is visible to all other threads
		// thread 0 of each block signals that it is done
		uint value = atomicInc(&count, gridDim.x);
		// determine if this block is the last block to be done
		isLastBlockDone = (value == (gridDim.x - 1));	// shared memory variable
	}
	// synchronize to make sure that each thread reads the correct value of isLastBlockDone
	__syncthreads();
	// the last block does the final summation
	if(isLastBlockDone)
	{
		VZero(partialSum[t]);		// reuse partial sum, t < blockDim.x
		
		if( t < gridDim.x  )		// max thread index is equal to gridDim.x
			// copy data from global memory to shared
			partialSum[t] = hlpArray[t];		

		VZero(partialSum[blockDim.x+t]);
		if( (blockDim.x+t) < gridDim.x )		
			partialSum[blockDim.x+t] = hlpArray[blockDim.x + t];			

		// begin summation, algorithm from lecture 13 Urbana, Illinois		
		for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
			__syncthreads();
			if( t < stride ) VVAdd(partialSum[t], partialSum[t + stride]);				
		}
		if(t == 0)	{
			// thread 0 of last block stores total sum to global memory
			// and resets count so next kernel calls work properly
			hlpArray[0] = partialSum[t];
			count = 0;
		}
	}	
}

// compute sum of squares of velocities
// it is stored in .x component of the 0 element of the hlpArray
__global__ void ComputeVvSumK( float3 *dv,	// array of velocities
						float3 *hlpArray,   // helper array, size of ceil(nMol / (2*blockDim.x)) is assumed
						real cmVelX)	// velocity of center of mass of Me should be excluded
{
	// we assume that each block at first computes partial sum of 2*blockDim.x elements
	extern __shared__ float3 partialSum[];	// size of 2*blockDim.x is assumed	
	uint stride, t = threadIdx.x;
	uint maxThreadIdx;

	int maxBlockIdx = floor(dparams.nMol*0.5f/blockDim.x);

	// for partially filled block define the index of the last thread
	if(maxBlockIdx == 0) maxThreadIdx = dparams.nMol;
	else
		maxThreadIdx = (uint) dparams.nMol % (2*blockDim.x);

	// copy data from global memory to shared
	partialSum[t] = dv[2*blockIdx.x*blockDim.x + t];
	partialSum[blockDim.x+t] = dv[(2*blockIdx.x + 1)*blockDim.x + t];

	// check if the thread is above the range then zero the sum
	// this causes divergent warp
	if( (blockIdx.x == maxBlockIdx) && (t >= maxThreadIdx ) ) 
		VZero(partialSum[t]);
	if( (blockIdx.x == maxBlockIdx) && ((blockDim.x+t) >= maxThreadIdx) ) 
		VZero(partialSum[blockDim.x+t]);

	// check if atom is above the range, then zero sums
	if( (2*blockIdx.x*blockDim.x + t) >= dparams.nMol){
		VZero(partialSum[t]);
	}
	if( ((2*blockIdx.x + 1)*blockDim.x + t) >= dparams.nMol){		
		VZero(partialSum[blockDim.x+t]);
	}	

	// if metal subtract velocity of center of mass	
	// here not threadIdx, but index of molecule!
	if( (dparams.iRegime == 2) && ( (2*blockIdx.x*blockDim.x + t) < dparams.nMolMe ) )
		partialSum[t].x = partialSum[t].x - cmVelX;		
	
	if( (dparams.iRegime == 2) && ( ( (2*blockIdx.x+1)*blockDim.x + t) < dparams.nMolMe ) )
		partialSum[blockDim.x+t].x = partialSum[blockDim.x+t].x - cmVelX;	

	// find square of velocity and store it in .x component
	partialSum[t].x = VLenSq (partialSum[t]);
	partialSum[blockDim.x + t].x = VLenSq (partialSum[blockDim.x + t]);

	// if metal scale by mass
	float m = dparams.massMe;
	// here not threadIdx, but index of molecule!
	if( (2*blockIdx.x*blockDim.x + t) < dparams.nMolMe )
		partialSum[t].x = m*partialSum[t].x;	
	if( ( (2*blockIdx.x+1)*blockDim.x + t) < dparams.nMolMe )
		partialSum[blockDim.x+t].x = m*partialSum[blockDim.x+t].x;
	
	// begin summation, algorithm from lecture 13 Urbana, Illinois
	// note that here all components are summed, but we're interested only in .x
	for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
		__syncthreads();
		if( t < stride ) VVAdd(partialSum[t], partialSum[t + stride]);
	}
	// in each block thread t == 0 contains partial sum
	// copy it to global memory (see p. 111 of programming guide)
	if( t == 0){
		hlpArray[blockIdx.x] = partialSum[t];		
		__threadfence();	// ensure that the result is visible to all other threads
		// thread 0 of each block signals that it is done
		uint value = atomicInc(&count, gridDim.x);
		// determine if this block is the last block to be done
		isLastBlockDone = (value == (gridDim.x - 1));	// shared memory variable
	}
	// synchronize to make sure that each thread reads the correct value of isLastBlockDone
	__syncthreads();
	// the last block does the final summation
	if(isLastBlockDone)
	{
		VZero(partialSum[t]);		// reuse partial sum, t < blockDim.x
		
		if( t < gridDim.x  )		// max thread index is equal to gridDim.x
			// copy data from global memory to shared
			partialSum[t] = hlpArray[t];		

		VZero(partialSum[blockDim.x+t]);
		if( (blockDim.x+t) < gridDim.x )		
			partialSum[blockDim.x+t] = hlpArray[blockDim.x + t];			

		// begin summation, algorithm from lecture 13 Urbana, Illinois		
		for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
			__syncthreads();
			if( t < stride ) VVAdd(partialSum[t], partialSum[t + stride]);				
		}
		if(t == 0)		{
			// thread 0  of last block stores total sum to global memory
			// and resets count so next kernel calls work properly
			hlpArray[0] = partialSum[t];
			count = 0;
		}
	}	
}

// find maximum of squares of velocities
// it is stored in .x component of the 0 element of the hlpArray
// this is also reduction, so the principle is the same as with summation,
// but instead of summation we compare elements
__global__ void ComputeVvMaxK( float3 *dv,	// array of velocities
						float3 *hlpArray )	// helper array, size of ceil(nMol / (2*blockDim.x)) is assumed
{
	// we assume that each block at first compares 2*blockDim.x elements
	extern __shared__ float3 partialSum[];	// size of 2*blockDim.x is assumed	
	uint stride, t = threadIdx.x;
	uint maxThreadIdx;

	int maxBlockIdx = floor(dparams.nMol*0.5f/blockDim.x);

	// for partially filled block define the index of the last thread
	if(maxBlockIdx == 0) maxThreadIdx = dparams.nMol;
	else
		maxThreadIdx = (uint) dparams.nMol % (2*blockDim.x);

	// copy data from global memory to shared
	partialSum[t] = dv[2*blockIdx.x*blockDim.x + t];
	partialSum[blockDim.x+t] = dv[(2*blockIdx.x + 1)*blockDim.x + t];

	// check if the thread is above the range, then zero the sum
	// this causes divergent warp
	if( (blockIdx.x == maxBlockIdx) && (t >= maxThreadIdx ) ) 
		VZero(partialSum[t]);
	if( (blockIdx.x == maxBlockIdx) && ((blockDim.x+t) >= maxThreadIdx) ) 
		VZero(partialSum[blockDim.x+t]);

	// check if atom is above the range, then zero sums
	if( (2*blockIdx.x*blockDim.x + t) >= dparams.nMol){
		VZero(partialSum[t]);
	}
	if( ((2*blockIdx.x + 1)*blockDim.x + t) >= dparams.nMol){		
		VZero(partialSum[blockDim.x+t]);
	}

	// find square of velocity and store it in .x component
	partialSum[t].x = VLenSq (partialSum[t]);
	partialSum[blockDim.x + t].x = VLenSq (partialSum[blockDim.x + t]);

	// begin comparison, algorithm from lecture 13 Urbana, Illinois
	// we're interested only in .x component
	for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
		__syncthreads();
		if( t < stride ) 
		{	// !here better to use atomic operation, but atomicMax() exists only for int
			if( partialSum[t].x < partialSum[t + stride].x)
				atomicExch(&partialSum[t].x, partialSum[t + stride].x);
		}
	}

	// in each block thread t == 0 contains maximum vv in .x component
	// copy it to global memory (see p. 111 of programming guide)
	if( t == 0)
	{
		hlpArray[blockIdx.x] = partialSum[t];		
		__threadfence();	// ensure that the result is visible to all other threads
		// thread 0 of each block signals that it is done
		uint value = atomicInc(&count, gridDim.x);
		// determine if this block is the last block to be done
		isLastBlockDone = (value == (gridDim.x - 1));	// shared memory variable
	}
	// synchronize to make sure that each thread reads the correct value of isLastBlockDone
	__syncthreads();
	// the last block does the final summation
	if(isLastBlockDone)
	{
		VZero(partialSum[t]);		// reuse partial sum, t < blockDim.x
		
		if( t < gridDim.x  )		// max thread index is equal to gridDim.x
			// copy data from global memory to shared
			partialSum[t] = hlpArray[t];		

		VZero(partialSum[blockDim.x+t]);
		if( (blockDim.x+t) < gridDim.x )		
			partialSum[blockDim.x+t] = hlpArray[blockDim.x + t];	
		// begin comparison, algorithm from lecture 13 Urbana, Illinois		
		for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
			__syncthreads();
			if( t < stride ){
				if( partialSum[t].x < partialSum[t + stride].x)
					atomicExch(&partialSum[t].x, partialSum[t + stride].x);
			}
		}
		if(t == 0){
			// thread 0  of last block stores maximum value to global memory
			// and resets count so next kernel calls work properly
			hlpArray[0] = partialSum[t];
			count = 0;
		}
	}	
}

// compute potential energy usum, it is = sum of .w components of dr elements
__global__ void ComputePotEnergyK( float4 *dr, float3 *hlpArray )	
{
	// we assume that each block at first computes partial sum of 2*blockDim.x elements
	extern __shared__ float3 partialSum[];	// size of 2*blockDim.x is assumed	
	uint stride, t = threadIdx.x;
	uint maxThreadIdx;

	int maxBlockIdx = floor(dparams.nMol*0.5f/blockDim.x);

	// for partially filled block define the index of the last thread
	if(maxBlockIdx == 0) maxThreadIdx = dparams.nMol;
	else
		maxThreadIdx = (uint) dparams.nMol % (2*blockDim.x);

	// copy data from global memory to shared
	// note the use of components, dr.w is copied to .x
	partialSum[t].x = dr[2*blockIdx.x*blockDim.x + t].w;
	partialSum[blockDim.x+t].x = dr[(2*blockIdx.x + 1)*blockDim.x + t].w;

	// check if the thread is above the range then zero the sum
	// this causes divergent warp
	if( (blockIdx.x == maxBlockIdx) && (t >= maxThreadIdx ) ) 
		VZero(partialSum[t]);
	if( (blockIdx.x == maxBlockIdx) && ((blockDim.x+t) >= maxThreadIdx) ) 
		VZero(partialSum[blockDim.x+t]);

	// check if atom is above the range, then zero sums
	if( (2*blockIdx.x*blockDim.x + t) >= dparams.nMol){
		VZero(partialSum[t]);
	}
	if( ((2*blockIdx.x + 1)*blockDim.x + t) >= dparams.nMol){		
		VZero(partialSum[blockDim.x+t]);
	}
	
	// begin summation, algorithm from lecture 13 Urbana, Illinois
	// note that here all components are summed, but we're interested only in .x
	for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
		__syncthreads();
		if( t < stride ) VVAdd(partialSum[t], partialSum[t + stride]);
	}

	// in each block thread t == 0 contains partial sum
	// copy it to global memory (see p. 111 of programming guide)
	if( t == 0)	{
		hlpArray[blockIdx.x] = partialSum[t];		
		__threadfence();	// ensure that the result is visible to all other threads
		// thread 0 of each block signals that it is done
		uint value = atomicInc(&count, gridDim.x);
		// determine if this block is the last block to be done
		isLastBlockDone = (value == (gridDim.x - 1));	// shared memory variable
	}
	// synchronize to make sure that each thread reads the correct value of isLastBlockDone
	__syncthreads();
	// the last block does the final summation
	if(isLastBlockDone)
	{
		VZero(partialSum[t]);		// reuse partial sum, t < blockDim.x
		
		if( t < gridDim.x  )		// max thread index is equal to gridDim.x
			// copy data from global memory to shared
			partialSum[t] = hlpArray[t];		

		VZero(partialSum[blockDim.x+t]);
		if( (blockDim.x+t) < gridDim.x )		
			partialSum[blockDim.x+t] = hlpArray[blockDim.x + t];			

		// begin summation, algorithm from lecture 13 Urbana, Illinois		
		for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
			__syncthreads();
			if( t < stride ) VVAdd(partialSum[t], partialSum[t + stride]);				
		}
		if(t == 0){
			// thread 0  of last block stores total sum to global memory
			// and resets count so next kernel calls work properly
			hlpArray[0] = partialSum[t];
			count = 0;
		}
	}	
}

// !note that in contrast to the upper functions
// in below functions we use dparams.nMolMe and not dparams.nMol
// because we are interested only in metal atoms

// compute coordinates of the center of mass of the nanoparticle (reduction)
__global__ void ComputeCenterOfMassK( float4 *dr,	// array of coordinates
						float3 *hlpArray )	// helper array, size of ceil(nMol / (2*blockDim.x)) is assumed
{
	// we assume that each block at first computes partial sum of 2*blockDim.x elements
	extern __shared__ float3 partialSum[];	// size of 2*blockDim.x is assumed	
	uint stride, t = threadIdx.x;
	uint maxThreadIdx;	

	int maxBlockIdx = floor(dparams.nMolMe*0.5f/blockDim.x);

	// for partially filled block define the index of the last thread
	// !note that here we consider only metal atoms!
	if(maxBlockIdx == 0) maxThreadIdx = dparams.nMolMe;
	else
		maxThreadIdx = (uint) dparams.nMolMe % (2*blockDim.x);

	// copy data from global memory to shared
	partialSum[t].x = dr[2*blockIdx.x*blockDim.x + t].x;
	partialSum[t].y = dr[2*blockIdx.x*blockDim.x + t].y;
	partialSum[t].z = dr[2*blockIdx.x*blockDim.x + t].z;
	partialSum[blockDim.x+t].x = dr[(2*blockIdx.x + 1)*blockDim.x + t].x;
	partialSum[blockDim.x+t].y = dr[(2*blockIdx.x + 1)*blockDim.x + t].y;
	partialSum[blockDim.x+t].z = dr[(2*blockIdx.x + 1)*blockDim.x + t].z;
	
	// check if the thread is above the range then zero the sum
	// this causes divergent warp
	if( (blockIdx.x == maxBlockIdx) && (t >= maxThreadIdx ) ) 
		VZero(partialSum[t]);
	if( (blockIdx.x == maxBlockIdx) && ((blockDim.x+t) >= maxThreadIdx) ) 
		VZero(partialSum[blockDim.x+t]);

	// check if this is not metal, then zero sums
	if( (2*blockIdx.x*blockDim.x + t) >= dparams.nMolMe)	{
		VZero(partialSum[t]);
	}
	if( ((2*blockIdx.x + 1)*blockDim.x + t) >= dparams.nMolMe)	{		
		VZero(partialSum[blockDim.x+t]);
	}

	// begin summation, algorithm from lecture 13 Urbana, Illinois	
	for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
		__syncthreads();
		if( t < stride ) VVAdd(partialSum[t], partialSum[t + stride]);
	}

	// in each block thread t == 0 contains partial sum
	// copy it to global memory (see p. 111 of programming guide)
	if( t == 0)	{
		hlpArray[blockIdx.x] = partialSum[t];		
		__threadfence();	// ensure that the result is visible to all other threads
		// thread 0 of each block signals that it is done
		uint value = atomicInc(&count, gridDim.x);
		// determine if this block is the last block to be done
		isLastBlockDone = (value == (gridDim.x - 1));	// shared memory variable
	}
	// synchronize to make sure that each thread reads the correct value of isLastBlockDone
	__syncthreads();
	// the last block does the final summation
	if(isLastBlockDone)
	{
		VZero(partialSum[t]);		// reuse partial sum, t < blockDim.x
		
		if( t < gridDim.x  )		// max thread index is equal to gridDim.x
			// copy data from global memory to shared
			partialSum[t] = hlpArray[t];		

		VZero(partialSum[blockDim.x+t]);
		if( (blockDim.x+t) < gridDim.x )		
			partialSum[blockDim.x+t] = hlpArray[blockDim.x + t];			

		// begin summation, algorithm from lecture 13 Urbana, Illinois		
		for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
			__syncthreads();
			if( t < stride ) VVAdd(partialSum[t], partialSum[t + stride]);				
		}
		if(t == 0)	{
			// thread 0  of last block stores total sum to global memory
			// and resets count so next kernel calls work properly
			hlpArray[0] = partialSum[t];
			count = 0;
		}
	}	
}

// compute velocity of the center of mass of the nanoparticle
// (sum of velocities of metal atoms)
__global__ void ComputeCmVelK( float3 *dv,	// array of velocities
						float3 *hlpArray )	// helper array, size of ceil(nMol / (2*blockDim.x)) is assumed
{
	// we assume that each block at first computes partial sum of 2*blockDim.x elements

	extern __shared__ float3 partialSum[];	// size of 2*blockDim.x is assumed	
	uint stride, t = threadIdx.x;
	uint maxThreadIdx;	
	
	int maxBlockIdx = floor(dparams.nMolMe*0.5f/blockDim.x);

	// for partially filled block define the index of the last thread
	if(maxBlockIdx == 0) maxThreadIdx = dparams.nMolMe;
	else
		maxThreadIdx = (uint) dparams.nMolMe % (2*blockDim.x);

	// copy data from global memory to shared
	partialSum[t] = dv[2*blockIdx.x*blockDim.x + t];
	partialSum[blockDim.x+t] = dv[(2*blockIdx.x + 1)*blockDim.x + t];

	// check if the thread is above the range then zero the sum
	// this causes divergent warp
	if( (blockIdx.x == maxBlockIdx) && (t >= maxThreadIdx ) ) 
		VZero(partialSum[t]);
	if( (blockIdx.x == maxBlockIdx) && ((blockDim.x+t) >= maxThreadIdx) ) 
		VZero(partialSum[blockDim.x+t]);

	// check if this is not metal, then zero sums
	if( (2*blockIdx.x*blockDim.x + t) >= dparams.nMolMe)	{
		VZero(partialSum[t]);
	}
	if( ((2*blockIdx.x + 1)*blockDim.x + t) >= dparams.nMolMe)	{		
		VZero(partialSum[blockDim.x+t]);
	}

	// begin summation, algorithm from lecture 13 Urbana, Illinois	
	for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
		__syncthreads();
		if( t < stride ) VVAdd(partialSum[t], partialSum[t + stride]);
	}

	// in each block thread t == 0 contains partial sum
	// copy it to global memory (see p. 111 of programming guide)
	if( t == 0)	{
		hlpArray[blockIdx.x] = partialSum[t];		
		__threadfence();	// ensure that the result is visible to all other threads
		// thread 0 of each block signals that it is done
		uint value = atomicInc(&count, gridDim.x);
		// determine if this block is the last block to be done
		isLastBlockDone = (value == (gridDim.x - 1));	// shared memory variable
	}
	// synchronize to make sure that each thread reads the correct value of isLastBlockDone
	__syncthreads();
	// the last block does the final summation
	if(isLastBlockDone)
	{
		VZero(partialSum[t]);		// reuse partial sum, t < blockDim.x
		
		if( t < gridDim.x  )		// max thread index is equal to gridDim.x
			// copy data from global memory to shared
			partialSum[t] = hlpArray[t];		

		VZero(partialSum[blockDim.x+t]);
		if( (blockDim.x+t) < gridDim.x )		
			partialSum[blockDim.x+t] = hlpArray[blockDim.x + t];			

		// begin summation, algorithm from lecture 13 Urbana, Illinois		
		for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
			__syncthreads();
			if( t < stride ) VVAdd(partialSum[t], partialSum[t + stride]);				
		}
		if(t == 0)	{
			// thread 0  of last block stores total sum to global memory
			// and resets count so next kernel calls work properly
			hlpArray[0] = partialSum[t];
			count = 0;
		}
	}	
}

// evaluate dimensions of the nanoparticle
// we find maximum or minimum values of coordinates 
// in corresponding directions and the difference between them on host
__global__ void ComputeParticleSizeK( float4 *dr,	// array of coordinates
								float3 *hlpArray,	// helper array, size of ceil(nMol / (2*blockDim.x)) is assumed
								int		min_max )	// if == 0 find minimum value, 1 - maximum
{
	// we assume that each block at first compares 2*blockDim.x elements
	extern __shared__ float3 partialSum[];	// size of 2*blockDim.x is assumed
	
	uint stride, t = threadIdx.x;
	uint maxThreadIdx;

	// consider only metal atoms
	int maxBlockIdx = floor(dparams.nMolMe*0.5f/blockDim.x);

	// for partially filled block define the index of the last thread
	if(maxBlockIdx == 0) maxThreadIdx = dparams.nMolMe;
	else
		maxThreadIdx = (uint) dparams.nMolMe % (2*blockDim.x);

	// copy data from global memory to shared
	partialSum[t].x = dr[2*blockIdx.x*blockDim.x + t].x;
	partialSum[t].y = dr[2*blockIdx.x*blockDim.x + t].y;
	partialSum[t].z = dr[2*blockIdx.x*blockDim.x + t].z;
	partialSum[blockDim.x+t].x = dr[(2*blockIdx.x + 1)*blockDim.x + t].x;
	partialSum[blockDim.x+t].y = dr[(2*blockIdx.x + 1)*blockDim.x + t].y;
	partialSum[blockDim.x+t].z = dr[(2*blockIdx.x + 1)*blockDim.x + t].z;

	// check if the thread is above the range, then zero the sum
	// this causes divergent warp
	if( (blockIdx.x == maxBlockIdx) && (t >= maxThreadIdx ) ) 
		VZero(partialSum[t]);
	if( (blockIdx.x == maxBlockIdx) && ((blockDim.x+t) >= maxThreadIdx) ) 
		VZero(partialSum[blockDim.x+t]);

	// check if this is not metal, then zero sums
	if( (2*blockIdx.x*blockDim.x + t) >= dparams.nMolMe){
		VZero(partialSum[t]);
	}
	if( ((2*blockIdx.x + 1)*blockDim.x + t) >= dparams.nMolMe){		
		VZero(partialSum[blockDim.x+t]);
	}

	// begin comparison, algorithm from lecture 13 Urbana, Illinois
	// .x component
	for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
		__syncthreads();
		if( t < stride ) 
		{	
			if(min_max == 0){	// find minimum value
				if( partialSum[t].x > partialSum[t + stride].x)
					atomicExch(&partialSum[t].x, partialSum[t + stride].x);				
			}
			else if(min_max == 1){	// find maximum value
				if( partialSum[t].x < partialSum[t + stride].x)
					atomicExch(&partialSum[t].x, partialSum[t + stride].x);				
			}
		}
	}
	// .y component
	for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
		__syncthreads();
		if( t < stride ) 
		{	
			if(min_max == 0){	// find minimum value
				if( partialSum[t].y > partialSum[t + stride].y)
					atomicExch(&partialSum[t].y, partialSum[t + stride].y);				
			}
			else if(min_max == 1){	// find maximum value
				if( partialSum[t].y < partialSum[t + stride].y)
					atomicExch(&partialSum[t].y, partialSum[t + stride].y);				
			}
		}
	}
	// .z component
	for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
		__syncthreads();
		if( t < stride ) 
		{	
			if(min_max == 0){	// find minimum value
				if( partialSum[t].z > partialSum[t + stride].z)
					atomicExch(&partialSum[t].z, partialSum[t + stride].z);				
			}
			else if(min_max == 1){	// find maximum value
				if( partialSum[t].z < partialSum[t + stride].z)
					atomicExch(&partialSum[t].z, partialSum[t + stride].z);				
			}
		}
	}

	// in each block thread t == 0 contains maximum value
	// copy it to global memory (see p. 111 of programming guide)
	if( t == 0)
	{
		hlpArray[blockIdx.x] = partialSum[t];		
		__threadfence();	// ensure that the result is visible to all other threads
		// thread 0 of each block signals that it is done
		uint value = atomicInc(&count, gridDim.x);
		// determine if this block is the last block to be done
		isLastBlockDone = (value == (gridDim.x - 1));	// shared memory variable
	}
	// synchronize to make sure that each thread reads the correct value of isLastBlockDone
	__syncthreads();
	// the last block does the final summation
	if(isLastBlockDone)
	{
		VZero(partialSum[t]);		// reuse partial sum, t < blockDim.x
		
		if( t < gridDim.x  )		// max thread index is equal to gridDim.x
			// copy data from global memory to shared
			partialSum[t] = hlpArray[t];		

		VZero(partialSum[blockDim.x+t]);
		if( (blockDim.x+t) < gridDim.x )		
			partialSum[blockDim.x+t] = hlpArray[blockDim.x + t];	
		// begin comparison, algorithm from lecture 13 Urbana, Illinois
		// .x component
		for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
			__syncthreads();
			if( t < stride ){
				if(min_max == 0){	// find minimum value
					if( partialSum[t].x > partialSum[t + stride].x)
						atomicExch(&partialSum[t].x, partialSum[t + stride].x);					
				}
				else if(min_max == 1){	// find maximum value
					if( partialSum[t].x < partialSum[t + stride].x)
						atomicExch(&partialSum[t].x, partialSum[t + stride].x);					
				}
			}
		}
		// .y component
		for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
			__syncthreads();
			if( t < stride ) 
			{	
				if(min_max == 0){	// find minimum value
					if( partialSum[t].y > partialSum[t + stride].y)
						atomicExch(&partialSum[t].y, partialSum[t + stride].y);				
				}
				else if(min_max == 1){	// find maximum value
					if( partialSum[t].y < partialSum[t + stride].y)
						atomicExch(&partialSum[t].y, partialSum[t + stride].y);				
				}
			}
		}
		// .z component
		for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
			__syncthreads();
			if( t < stride ) 
			{	
				if(min_max == 0){	// find minimum value
					if( partialSum[t].z > partialSum[t + stride].z)
						atomicExch(&partialSum[t].z, partialSum[t + stride].z);				
				}
				else if(min_max == 1){	// find maximum value
					if( partialSum[t].z < partialSum[t + stride].z)
						atomicExch(&partialSum[t].z, partialSum[t + stride].z);				
				}
			}
		}
		if(t == 0){
			// thread 0 of last block stores maximum value to global memory
			// and resets count so next kernel calls work properly
			hlpArray[0] = partialSum[t];
			count = 0;
		}
	}	// end if(isLastBlockDone)
}

// compute total force acting on the nanoparticle (reduction)
__global__ void ComputeNetForceK( real *hlpArray )	// array of accelerations and helper array
{
	// we assume that each block at first computes partial sum of 2*blockDim.x elements
	extern __shared__ float3 partialSum[];	// size of 2*blockDim.x is assumed	
	uint stride, t = threadIdx.x;
	uint maxThreadIdx;	

	// note that we consider only metal atoms
	int maxBlockIdx = floor(dparams.nMolMe*0.5f/blockDim.x);

	// for partially filled block define the index of the last thread
	// !note that here we consider only metal atoms!
	if(maxBlockIdx == 0) maxThreadIdx = dparams.nMolMe;
	else
		maxThreadIdx = (uint) dparams.nMolMe % (2*blockDim.x);

	// copy data from global memory to shared
	// !check index to avoid memory problems!
	if( (2*blockIdx.x*blockDim.x + t) < dparams.nMolMe)
		partialSum[t].x = hlpArray[2*blockIdx.x*blockDim.x + t];
	if( ((2*blockIdx.x + 1)*blockDim.x + t) < dparams.nMolMe)
		partialSum[blockDim.x+t].x = hlpArray[(2*blockIdx.x + 1)*blockDim.x + t];
	
	// check if the thread is above the range then zero the sum
	// this causes divergent warp
	if( (blockIdx.x == maxBlockIdx) && (t >= maxThreadIdx ) ) 
		VZero(partialSum[t]);
	if( (blockIdx.x == maxBlockIdx) && ((blockDim.x+t) >= maxThreadIdx) ) 
		VZero(partialSum[blockDim.x+t]);

	// check if this is not metal, then zero sums
	if( (2*blockIdx.x*blockDim.x + t) >= dparams.nMolMe){
		VZero(partialSum[t]);
	}
	if( ((2*blockIdx.x + 1)*blockDim.x + t) >= dparams.nMolMe){		
		VZero(partialSum[blockDim.x+t]);
	}

	// begin summation, algorithm from lecture 13 Urbana, Illinois	
	for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
		__syncthreads();
		if( t < stride ) VVAdd(partialSum[t], partialSum[t + stride]);
	}

	// in each block thread t == 0 contains partial sum
	// copy it to global memory (see p. 111 of programming guide)
	if( t == 0)	{
		hlpArray[blockIdx.x] = partialSum[t].x;		
		__threadfence();	// ensure that the result is visible to all other threads
		// thread 0 of each block signals that it is done
		uint value = atomicInc(&count, gridDim.x);
		// determine if this block is the last block to be done
		isLastBlockDone = (value == (gridDim.x - 1));	// shared memory variable
	}
	// synchronize to make sure that each thread reads the correct value of isLastBlockDone
	__syncthreads();
	// the last block does the final summation
	if(isLastBlockDone)
	{
		VZero(partialSum[t]);		// reuse partial sum, t < blockDim.x
		
		if( t < gridDim.x  )		// max thread index is equal to gridDim.x
			// copy data from global memory to shared
			partialSum[t].x = hlpArray[t];		

		VZero(partialSum[blockDim.x+t]);
		if( (blockDim.x+t) < gridDim.x )		
			partialSum[blockDim.x+t].x = hlpArray[blockDim.x + t];			

		// begin summation, algorithm from lecture 13 Urbana, Illinois		
		for(stride = blockDim.x; stride >= 1; stride = stride >> 1) {
			__syncthreads();
			if( t < stride ) VVAdd(partialSum[t], partialSum[t + stride]);				
		}
		if(t == 0)	{
			// thread 0  of last block stores total sum to global memory
			// and resets count so next kernel calls work properly
			hlpArray[0] = partialSum[t].x;
			count = 0;
		}
	}	
}

// compute radial distribution function
__global__ void EvalRdfK (float4 *dr,			// coordinates of molecules (global)							  
						  int *CELL,			// indexes of molecules (global)
						  uint *histRdf,			// histogram for rdf
						  int countRdf)			// number of measurements						  
{		
	int3 cc, m2v, vOff[] = OFFSET_VALS_LONG;	// 3D indexes of the cells
	int i, j, offset, n;						// counters
	int C;										// 1D cell indexes	
	float3 shift;								// variable for handling boundary conditions
	float4 A, deltaR;							// radus-vectors
	real rr;				// square of distance between atoms
	int indexRdf;			// index in the histogram

	// define 3D index of the current cell, blockIdx.x is 1D index of the cell
	cc.z = blockIdx.x / (dparams.cells.x * dparams.cells.y);
	cc.y = (blockIdx.x - cc.z * dparams.cells.x * dparams.cells.y) / dparams.cells.x;
	cc.x = blockIdx.x - (cc.z*dparams.cells.y + cc.y)*dparams.cells.x;
	
// here begins implementation of the Algorithm 4 from Anderson
	n = blockIdx.x * blockDim.x + threadIdx.x;

	// zero histogram for rdf if the first time
	if( (countRdf == 0) && (n < dparams.sizeHistRdf) )
		histRdf[n] = 0.f;	

	if(n < dparams.maxMol)
		i = CELL[blockIdx.x * blockDim.x + threadIdx.x];// step 1, get particle index
	else i = -1;	
	if(i != -1)	{									// added by me to avoid memory problems
		A = dr[i];									// step 3			
	}
	// begin step 4
	for (offset = 0; offset < N_OFFSET_LONG; offset ++)	// loop over all 27 neighboring cells 
	{
		VAdd (m2v, cc, vOff[offset]);			// find 3D index of the neighboring cell
		
		shift.x = 0.f;							// zero shift for boundary conditions
		shift.y = 0.f;
		shift.z = 0.f;

		// apply boundary conditions
		// !type manually instead of using macroses, because they do not work!
		// x
		if (m2v.x >= dparams.cells.x) { 
			m2v.x = 0; 
			shift.x = dparams.region.x; 
		} else if (m2v.x < 0) {
			m2v.x = dparams.cells.x - 1;
			shift.x = - dparams.region.x;
		}
		// y
		if (m2v.y >= dparams.cells.y) { 
			m2v.y = 0; 
			shift.y = dparams.region.y; 
		} else if (m2v.y < 0) {
			m2v.y = dparams.cells.y - 1;
			shift.y = - dparams.region.y;
		}
		// z
		if (m2v.z >= dparams.cells.z) { 
			m2v.z = 0; 
			shift.z = dparams.region.z; 
		} else if (m2v.z < 0) {
			m2v.z = dparams.cells.z - 1;
			shift.z = - dparams.region.z;
		}

		C = VLinear (m2v, dparams.cells);			// find 1D index of the neighboring cell
	// end step 4

		__syncthreads();							// step 5
		K[threadIdx.x] = CELL[C * blockDim.x + threadIdx.x];	// step 6
		if(K[threadIdx.x] != -1)					// added by me, to avoid memory error
			B[threadIdx.x] = dr[ K[threadIdx.x] ];	// step 7
		else
			VZero(B[threadIdx.x]);					// added by me
		__syncthreads();							// step 8		
		
		if( i != -1)	{							// step 9
			for(j = 0; j < blockDim.x; j++)	
			{
				// step 10, loop over atoms from the current neighboring cell C
				if( K[j] != -1 )	{				// steps 11 - 13,	
					
					deltaR.x = A.x - B[j].x;		// step 14
					deltaR.y = A.y - B[j].y;
					deltaR.z = A.z - B[j].z;
					
					deltaR.x = deltaR.x - shift.x;	// step 15, boundary conditions
					deltaR.y = deltaR.y - shift.y;
					deltaR.z = deltaR.z - shift.z;

					rr = deltaR.x*deltaR.x + deltaR.y*deltaR.y + deltaR.z*deltaR.z;
					// !consider only metal atoms! step 16
					if ( ( rr < dparams.rangeRdf*dparams.rangeRdf) && (i != K[j]) &&
						(i < dparams.nMolMe) && (K[j]< dparams.nMolMe))	
					{						
						indexRdf = sqrt(rr)*dparams.intervalRdf;// step 17
						atomicAdd(&histRdf[indexRdf], 1);		// step 18
					}											// step 19
				}	// end if( K[j] != -1 )
			}													// step 20 
		}														// step 21	
	} // step 22 end for (offset = 0; offset < N_OFFSET_LARGE; offset ++) 		
}


////////////////////////////////////////////////
// wrappers - W at the end of the function name
////////////////////////////////////////////////
extern "C"	// can be deleted, but then also in SurfaceGrowth.h
{
////////////////////////////
// wrappers calling kernels
////////////////////////////
	

// calls computational kernels when OpenGL is not used
char* DoComputationsW(float4 *hr, float3 *hv, float3 *ha, SimParams *hparams, 
					 FILE *fResults, TCHAR *szPdbPath)
{	
	// pointers to host memory
	uint	*hHistRdf = 0;
	if(hparams->bRdf != 0)
		AllocMem(hHistRdf, hparams->sizeHistRdf, uint);

	// host variables for diffusion
	int nb;
	TBuf *tBuf;
	real *rrDiffuseAv;
	FILE *fileDiffuse = NULL;
	if(hparams->iRegime == 2)	// if shear then measure diffusion
	{
		AllocMem(tBuf, hparams->nBuffDiffuse, TBuf);
		AllocMem(rrDiffuseAv, hparams->nValDiffuse, real);
		for(nb = 0; nb < hparams->nBuffDiffuse; nb++)
			AllocMem(tBuf[nb].rrDiffuse, hparams->nValDiffuse, real);
		InitDiffusion(tBuf, rrDiffuseAv, hparams);
		// open file for diffusion
		if( (fileDiffuse = _tfopen(hparams->szDiffusePath, TEXT("w"))) == NULL ){
			lstrcpy(szPdbPath, "Cannot open diffuse file!");
			return szPdbPath;
		}
	}

	// pointers to device memory
	float3	*dv,			// velocities
			*da,			// accelerations			
			*hlpArray;		// helper array
	float4	*dr;			// positions

	real	*dcarbonForce = 0;	// forces acting on metal atoms from carbon atoms (in x direction)

	real	*rho;			// electron density for eam

	int		*CELL;			// indexes of particles for cells
	int		*NN,			// number of neighbors for each particle		
			*NBL;			// neighbors list, i.e. indeces of neighboring particles
	uint	*molsInCells;	// number of atoms in each cell, size is = number of cells

	uint	*histRdf = 0;	// histogram for rdf

	// number of blocks for summing and the size of the array hlpArray;
	// each block contains 512 threads and processes 1024 elements;
	// because the last block computes total sum of grid elements,
	// so grid could not be greater than 1024, and hence nMol <= 1024 * 1024 = 1048576
	uint grid = (uint) ceil((float)hparams->nMol / (1024) );
	uint block = 512;		// threads per block for summation

	dim3 dimBlock(hparams->blockSize, 1, 1);	// number of threads	
	// define number of blocks as in Anderson
	dim3 dimGrid(hparams->gridSize);	

	float3	vSum;			// total impulse
	real	vvSum = 0.f, vvMax = 0.f, uSum = 0.f;

	// tribological properties
	float3	centerOfMass, frictForce, cmVel;
	float3	particleSize, particleSizeMin;	// dimensions of the nanoparticle

	int iBckup = 1;		// for choosing of the backup file
	
	// for error handling
	hipError_t error;

	float hTime, hTimeTotal;	// time of one time step and of the complete run
	hipEvent_t start, stop, totalStart, totalStop;

// begin memory allocation	
	hipMalloc(&dr, hparams->nMol * sizeof(float4));
	// also allocate memory for velocities and accelerations	
	hipMalloc(&dv, hparams->nMol * sizeof(float3));	
	hipMalloc(&da, hparams->nMol * sizeof(float3));

	// helper array for computing system properties, total impulse, energy, etc.
	// its size is the size of the grid of blocks for summing
	hipMalloc(&hlpArray, grid * sizeof(float3));

	// for eam
	hipMalloc(&rho, hparams->nMolMe * sizeof(real));

	// allocate memory for particle indexes and neighbor list
	// !note (hparams->maxMol+1) to avoid subtle bug!
	hipMalloc(&CELL, (hparams->maxMol+1) * sizeof(int));
	// number of neighbors
	hipMalloc(&NN, hparams->nMol * sizeof(int));
	// indexes of neighboring atoms
	hipMalloc(&NBL, hparams->nMol * hparams->iNebrMax * sizeof(int));
	// number of atoms in each cell
	hipMalloc(&molsInCells, VProd(hparams->cells) * sizeof(uint));	

	if( (hparams->iRegime != 0) && (hparams->nMolMe != 0) )	// allocate memory for friction force
		hipMalloc(&dcarbonForce, hparams->nMolMe * sizeof(real));
	// if needed allocate memory for rdf
	if( hparams->bRdf != 0 )
		hipMalloc(&histRdf, hparams->sizeHistRdf * sizeof(int));

	// check errors	
	error = hipGetLastError();
	if( error != hipSuccess) {
		hipDeviceReset();
		const char* errorString = hipGetErrorString (error);
		lstrcpy(szPdbPath, "Problems with memory allocation! Exception: ");
		lstrcat(szPdbPath, errorString);		
		return szPdbPath;
	}
// end memory allocation

	// copy data from host to device
	hipMemcpy(dr, hr, hparams->nMol*sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(dv, hv, hparams->nMol*sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpy(da, ha, hparams->nMol*sizeof(float3), hipMemcpyHostToDevice);

	// if needed, make the first coordinate snapshot
	if( hparams->bPdb != 0 ) {		
		CreatePdbFile(szPdbPath, hparams, hr);			
	}		
	
	// initialize variables for computing of one timestep
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// for total time
	hipEventCreate(&totalStart);
	hipEventCreate(&totalStop);

// begin computation of one time step
	while(hparams->moreCycles)
	{		
		hipEventRecord(start, 0);	// record start time
		hipEventRecord(totalStart, 0);
		
		++ hparams->stepCount;	// increment step count

// code for insertion of atoms (for surface growth regime)
	// before equilibration atoms are not deposited
	if( (hparams->iRegime == 1) && (hparams->stepCount <= hparams->stepEquil) )
	{	
		int nMolToDeposit = 0;		
		InsertAtomsK<<< dimGrid, dimBlock >>>
			( dr, dv, hparams->nMolDeposited, nMolToDeposit );		
	}
	if( (hparams->iRegime == 1) && (hparams->stepCount > hparams->stepEquil) )
	{
		if(hparams->nMolDeposited < hparams->nMolMe)
		{
		int nMolToDeposit = 0;
		if((hparams->stepCount-hparams->stepEquil) % hparams->stepDeposit == 0)
				nMolToDeposit = hparams->nMolToDeposit;
			InsertAtomsK<<< dimGrid, dimBlock >>>
				( dr, dv, hparams->nMolDeposited, nMolToDeposit );
			// enlarge the number of deposited atoms
			hparams->nMolDeposited += nMolToDeposit;			
		}
	}
	// check errors	
	error = hipGetLastError();
	if( error != hipSuccess) {
		hipDeviceReset();
		const char* errorString = hipGetErrorString (error);
		lstrcpy(szPdbPath, "Problems with insertion of atoms! Exception: ");
		lstrcat(szPdbPath, errorString);		
		return szPdbPath;
	}
// end code for insertion of atoms
		
		LeapfrogStepK<<< dimGrid, dimBlock >>> ( 1, dr, dv, da );
		ApplyBoundaryCondK<<< dimGrid, dimBlock >>> ( dr );

		// check errors	
		error = hipGetLastError();
		if( error != hipSuccess) {
			hipDeviceReset();
			const char* errorString = hipGetErrorString (error);
			lstrcpy(szPdbPath, "Problems with the 1st part of Verlet! Exception: ");
			lstrcat(szPdbPath, errorString);		
			return szPdbPath;
		}

		if (hparams->nebrNow) {
			hparams->nebrNow = 0;
			hparams->dispHi = 0.f;
			// fill cells with -1 (empty particles)
			// !note (hparams->maxMol+1) to avoid subtle bug with index 0
			hipMemset(CELL, -1, (hparams->maxMol+1) * sizeof(int));
			// fill number of neighbors with 0
			hipMemset(NN, 0, hparams->nMol * sizeof(int));
			// fill neighbor list with -1
			hipMemset(NBL, -1, hparams->nMol * hparams->iNebrMax * sizeof(int)); 
			// fill number of atoms in each cell by 0
			hipMemset(molsInCells, 0, VProd(hparams->cells) * sizeof(uint));
			// define cells of atoms
			BinAtomsIntoCellsK<<< dimGrid, dimBlock >>> (dr, CELL, molsInCells);
			// check errors	
			error = hipGetLastError();
			if( error != hipSuccess) {
				hipDeviceReset();
				const char* errorString = hipGetErrorString (error);
				lstrcpy(szPdbPath, "Problems with building of cells! Exception: ");
				lstrcat(szPdbPath, errorString);		
				return szPdbPath;
			}

			BuildNebrListK<<< dimGrid, dimBlock >>>	(dr, CELL, NN, NBL);
			// check errors
			error = hipGetLastError();
			if( error != hipSuccess) {
				hipDeviceReset();
				const char* errorString = hipGetErrorString (error);
				lstrcpy(szPdbPath, "Problems with building of neigbor list! Exception: ");
				lstrcat(szPdbPath, errorString);		
				return szPdbPath;
			}
		}		
		if( hparams->nMolMe != 0)
			EamComputeRhoK<<< dimGrid, dimBlock >>>(rho, dr, NN, NBL);
		
		// compute and save in the dcarbonForce forces acting from C on Me
		ComputeForcesK<<< dimGrid, dimBlock >>> (da, dr, NN, NBL, rho, dcarbonForce);
		// check errors
		if( error != hipSuccess) {
			hipDeviceReset();
			const char* errorString = hipGetErrorString (error);
			lstrcpy(szPdbPath, "Problems with foce evaluation! Exception: ");
			lstrcat(szPdbPath, errorString);		
			return szPdbPath;
		}

		if( dcarbonForce != 0 )
		{
			ComputeNetForceK<<< grid,		// number of blocks <= 1024
				block,						// number of threads
				2*block*sizeof(float3)		// memory for dynamic array in shared memory
				>>>(dcarbonForce);
			// copy force on host
			hipMemcpy(&frictForce, dcarbonForce, sizeof(real), hipMemcpyDeviceToHost);
		}
		else frictForce.x = 0.f;

		LeapfrogStepK<<< dimGrid, dimBlock >>> ( 2, dr, dv, da );	

// kernels, that form EvalProps - evaluate properties
// begin compute tribological properties
		if(hparams->nMolMe != 0)			// avoid bad values without metal atoms
		{
			// compute coordinate of center of mass of the nanoparticle
			ComputeCenterOfMassK<<< grid,	// number of blocks <= 1024
				block,						// number of threads
				2*block*sizeof(float3)		// memory for dynamic array in shared memory
				>>>(dr, hlpArray);
			// copy center of mass on host
			hipMemcpy(&centerOfMass, hlpArray, sizeof(float3), hipMemcpyDeviceToHost);

			// compute velocity of center of mass of the nanoparticle
			ComputeCmVelK<<< grid,			// number of blocks <= 1024
				block,						// number of threads
				2*block*sizeof(float3)		// memory for dynamic array in shared memory
				>>>(dv, hlpArray);
			// copy velocity on host
			hipMemcpy(&cmVel, hlpArray, sizeof(float3), hipMemcpyDeviceToHost);

			// compute dimensions of the nanoparticle
			// compute minimum radius-vector
			ComputeParticleSizeK<<< grid,	// number of blocks <= 1024
				block,						// number of threads
				2*block*sizeof(float3)		// memory for dynamic array in shared memory
				>>>(dr, hlpArray, 0);
			// copy minimum radius vector on host
			hipMemcpy(&particleSizeMin, hlpArray, sizeof(float3), hipMemcpyDeviceToHost);
			// compute maximum radius-vector
			ComputeParticleSizeK<<< grid,				// number of blocks <= 1024
				block,						// number of threads
				2*block*sizeof(float3)		// memory for dynamic array in shared memory
				>>>(dr, hlpArray, 1);
			// copy maximum radius-vector on host
			hipMemcpy(&particleSize, hlpArray, sizeof(float3), hipMemcpyDeviceToHost);
			// find difference particleSize = particleSize - particleSizeMin;
			particleSize.x = particleSize.x - particleSizeMin.x;
			particleSize.y = particleSize.y - particleSizeMin.y;
			particleSize.z = particleSize.z - particleSizeMin.z;

			// check errors
			error = hipGetLastError();
			if( error != hipSuccess) {
				hipDeviceReset();
				const char* errorString = hipGetErrorString (error);
				lstrcpy(szPdbPath, "Problems with tribological properties! Exception: ");
				lstrcat(szPdbPath, errorString);		
				return szPdbPath;
			}

			// save properties
			// consider only x component because shear is in x direction
			hparams->centerOfMass.val = centerOfMass.x / hparams->nMolMe;
			hparams->cmVel.val = cmVel.x / hparams->nMolMe;
			hparams->frictForce.val = frictForce.x;
			hparams->particleSize.x = particleSize.x;
			hparams->particleSize.y = particleSize.y;
			hparams->particleSize.z = particleSize.z;

			// calculate diffusion if shear regime
			if(hparams->iRegime == 2) {
				if( (hparams->stepCount > hparams->stepEquil) &&
					( (hparams->stepCount - hparams->stepEquil) % hparams->stepDiffuse == 0) )
				{
					centerOfMass.x /= hparams->nMolMe;
					centerOfMass.y /= hparams->nMolMe;
					centerOfMass.z /= hparams->nMolMe;
					EvalDiffusion(tBuf, rrDiffuseAv, fileDiffuse, hparams, centerOfMass);
				}
			}

		}	// end if(hparams->nMolMe != 0)
		else
		{
			hparams->centerOfMass.val = 0.f;
			hparams->cmVel.val = 0.f;
			hparams->frictForce.val = 0.f;
			hparams->particleSize.x = 0.f;
			hparams->particleSize.y = 0.f;
			hparams->particleSize.z = 0.f;
		}
// end compute tribological properties

		// compute total impulse
		ComputeVSumK<<< grid,						// number of blocks <= 1024
			block,						// number of threads
			2*block*sizeof(float3)		// memory for dynamic array in shared memory
			>>>(dv, hlpArray);
		// copy total impulse on host
		hipMemcpy(&vSum, hlpArray, sizeof(float3), hipMemcpyDeviceToHost);

		// find maximum of squares of velocities
		ComputeVvMaxK<<< grid,						// number of blocks <= 1024
						block,						// number of threads
						2*block*sizeof(float3)		// memory for dynamic array in shared memory
					>>>(dv, hlpArray);
		// copy maximum of squares of velocities on host
		hipMemcpy(&vvMax, &hlpArray[0].x, sizeof(float), hipMemcpyDeviceToHost);

		// compute sum of squares of velocities
		ComputeVvSumK<<< grid,						// number of blocks <= 1024
						block,						// number of threads
						2*block*sizeof(float3)		// memory for dynamic array in shared memory
						>>>(dv, hlpArray, hparams->cmVel.val);
		// copy sum of squares of velocities on host
		hipMemcpy(&vvSum, &hlpArray[0].x, sizeof(float), hipMemcpyDeviceToHost);

		// apply Berendsen thermostat
		// for shear apply thermostat to all atoms after stepEquil,
		// for metal only during step cool, for carbon - till the end
		if( (hparams->iRegime == 2) && (hparams->stepCount > hparams->stepEquil)
			&& (hparams->stepCount % hparams->stepThermostat == 0))
			ApplyBerendsenThermostat<<< dimGrid, dimBlock >>> 
			( dv, &hlpArray[0].x, hparams->stepCount );	// here if step > cool Me is not thermostatted	

		// for SG apply thermostat to carbon atoms during all simulation
		if( (hparams->iRegime == 1)	&& (hparams->stepCount % hparams->stepThermostat == 0))
			ApplyBerendsenThermostat<<< dimGrid, dimBlock >>> 
			( dv, &hlpArray[0].x, hparams->stepCount );
		
		// compute potential energy
		ComputePotEnergyK<<< grid, block, 2*block*sizeof(float3) >>>(dr, hlpArray);
		// copy potential energy on host
		hipMemcpy(&uSum, &hlpArray[0].x, sizeof(float), hipMemcpyDeviceToHost);
		// check errors
		error = hipGetLastError();
		if( error != hipSuccess) {
			hipDeviceReset();
			const char* errorString = hipGetErrorString (error);
			lstrcpy(szPdbPath, "Problems with evaluation of properties! Exception: ");
			lstrcat(szPdbPath, errorString);		
			return szPdbPath;
		}

		// see whether the building of the list is ripe
		hparams->dispHi += sqrt (vvMax) * hparams->deltaT;	
		if (hparams->dispHi > 0.5f * hparams->rNebrShell) hparams->nebrNow = 1;
		hparams->kinEnergy.val = 0.5f * vvSum /	hparams->nMol;	// compute kinetic energy
		hparams->potEnergy.val = uSum / hparams->nMol;			// potential energy
		hparams->totEnergy.val = hparams->kinEnergy.val + hparams->potEnergy.val;
		VCopy(hparams->vSum, vSum);	// copy impulse

// begin apply shear
	if( (hparams->iRegime == 2) && (hparams->nMolMe != 0) )
	{
		if( hparams->stepCount > (hparams->stepEquil + hparams->stepCool) )
		{
			if( hparams->cmVel.val < 0.005f )	// !adjust value of cmVel !
				hparams->shear += hparams->deltaF;

			// zero number of atoms to which shear is applied
			hipMemset(&molsInCells[0], 0, sizeof(uint));			
			ApplyShearK<<< dimGrid, dimBlock >>> ( dr, da, hparams->shear, hparams->centerOfMass.val, &molsInCells[0] );
			// copy number of sheared atoms
			uint hnOfShearedMol = 0;
			hipMemcpy(&hnOfShearedMol, &molsInCells[0], sizeof(uint), hipMemcpyDeviceToHost);
			// !compute total shear force			
			hparams->totalShear = hnOfShearedMol*hparams->shear;
		}
	}
// end apply shear

// begin compute rdf
	if( (hparams->bRdf != 0) && ( hparams->stepCount % hparams->stepRdf == 0 ) )
	{		
		EvalRdfK<<< dimGrid, dimBlock >>> (dr, CELL, histRdf, hparams->countRdf);		
		++hparams->countRdf;
		if(hparams->countRdf == hparams->limitRdf)
		{
			// copy rdf on host
			hipMemcpy(hHistRdf, histRdf, hparams->sizeHistRdf*sizeof(uint), hipMemcpyDeviceToHost);			
			PrintRdf(hparams, hHistRdf);
			hparams->countRdf = 0;
		}
	}
// end compute rdf

	hipEventRecord(stop, 0);		// record end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&hTime, start, stop);		

	hparams->oneStep.val = hTime;
// end EvalProps

	AccumProps (1, hparams);
	if (hparams->stepCount % hparams->stepAvg == 0) {
		AccumProps (2, hparams);
		if( hparams->bResult != 0 )
			PrintSummary (fResults, hparams);
		AccumProps (0, hparams);
	}		

	if( (hparams->stepCount % hparams->stepPdb == 0) && (hparams->bPdb!=0)) {
		// copy memory from device to host
		hipMemcpy(hr, dr, hparams->nMol*sizeof(float4), hipMemcpyDeviceToHost);
		CreatePdbFile(szPdbPath, hparams, hr);			
	}		
	
	if( hparams->stepCount >= hparams->stepLimit )		
		hparams->moreCycles = 0;	

// make backup if needed
	if( (hparams->bBckup) && (hparams->stepCount % hparams->stepBckup == 0) )
	{
		FILE *file = NULL;		
		// copy data from device to host
		// if stepBckup is dividable by stepPdb, then hr has already been copied
		if(hparams->stepBckup % hparams->stepPdb != 0)
			hipMemcpy(hr, dr, hparams->nMol*sizeof(float4), hipMemcpyDeviceToHost);
		hipMemcpy(hv, dv, hparams->nMol*sizeof(float3), hipMemcpyDeviceToHost);
		hipMemcpy(ha, da, hparams->nMol*sizeof(float3), hipMemcpyDeviceToHost);
		if(iBckup) {
			iBckup = 0;
			file = fopen(hparams->szBckup0, "w+b");
		}
		else {
			iBckup = 1;
			file = fopen(hparams->szBckup1, "w+b");
		}
		// write data to file		
		fwrite((void*)hparams, sizeof(SimParams), 1, file);
		fwrite((void*)hr, sizeof(float4), hparams->nMol, file);
		fwrite((void*)hv, sizeof(float3), hparams->nMol, file);
		fwrite((void*)ha, sizeof(float3), hparams->nMol, file);
		fclose(file);			
	}	// end if( (hparams->bBckup) &&	...
// end make backup
	
	// record total one step
	hipEventRecord(totalStop, 0);		// record end time
	hipEventSynchronize(totalStop);
	hipEventElapsedTime(&hTimeTotal, totalStart, totalStop);
	hparams->totalTime += hTimeTotal*0.001f;

	}	// end while(hparams->moreCycles)
// end computation of one time step	

	// print total time
	if( hparams->bResult != 0 )
		fprintf (fResults, "\nDuration of the simulation = %f s", hparams->totalTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipEventDestroy(totalStart);
	hipEventDestroy(totalStop);

	// cleanup
	hipFree(dr);	
	hipFree(dv);
	hipFree(da);
	hipFree(hlpArray);
	hipFree(rho);
	hipFree(CELL);
	hipFree(NN);
	hipFree(NBL);	
	hipFree(molsInCells);
	if( dcarbonForce != 0)	// if regime != 0
		hipFree(dcarbonForce);
	if((hparams->bRdf != 0) && (histRdf != 0) )
		hipFree(histRdf);
	// free host memory
	if( (hparams->bRdf != 0) && (hHistRdf != 0) )
		free(hHistRdf);
	// free buffers for diffusion
	if(hparams->iRegime == 2) {
		for(nb = 0; nb < hparams->nBuffDiffuse; nb++)
			if(tBuf[nb].rrDiffuse) free(tBuf[nb].rrDiffuse);
	}
	if(tBuf) free(tBuf);
	if(rrDiffuseAv) free(rrDiffuseAv);
	if(fileDiffuse != NULL)
		fclose(fileDiffuse);
		
	return 0;
}

// wrapper that initializes coordinates
const char* InitCoordsW(float4 *dr, float4 *hr, SimParams* hparams)	
{
	int i;
	hipMalloc(&dr, hparams->nMol * sizeof(float4));	// allocate device memory 
	if(hparams->iRegime == 0)	// if bulk, then only fcc lattice
	{
		uint max, middle, min;	// numbers of unit cells
		max = hparams->initUcell.x;
		middle = hparams->initUcell.y;
		min = hparams->initUcell.z;		
		
		// each thread handles one unit cell, and 4 atoms for fcc lattice
		dim3 dimBlock(1, 1, min);		// number of threads in the block (3D), min <= 512 cells
		// each block handles a strip of height min cells, length and width is 1 cell
		dim3 dimGrid(max, middle);		// grid size (number of blocks in 2D grid)
		// execute the kernel    
		InitFccCoordsK<<< dimGrid, dimBlock >>>(dr);
		// copy memory from device to host
		hipMemcpy(hr, dr, hparams->nMol*sizeof(float4), hipMemcpyDeviceToHost);
	}
	else if( hparams->iRegime == 1)	// if surface growth, generate random Me coordinates for Me
	{
		dim3 dimBlockCarbon(32, 1, 1);
		int numBlocks = (hparams->nMol-hparams->nMolMe) / dimBlockCarbon.x;
		dim3 dimGridCarbon(numBlocks, 1, 1);
		InitGrapheneCoordsK<<< dimGridCarbon, dimBlockCarbon >>>(dr);
		// copy memory from device to host here to avoid overwriting memory for metal
		hipMemcpy(hr, dr, hparams->nMol*sizeof(float4), hipMemcpyDeviceToHost);

		// generate random coordinates for metal atoms on host in advance
		for(i = 0; i < hparams->nMolMe; i++)
		{
			VRandRfloat4(&hr[i], hparams);
			hr[i].x = 0.5f*hparams->region.x*hr[i].x;
			hr[i].y = 0.5f*hparams->region.y*hr[i].y;
			hr[i].z = 1.5f*hparams->region.z + 0.5f*(i+1)*hparams->region.z;
			hr[i].w = 0.f;
		}
	} 
	else if( hparams->iRegime == 2)	// if shear, use Slab coords for metal
	{
		dim3 dimBlockCarbon(32, 1, 1);
		int numBlocks = (hparams->nMol-hparams->nMolMe) / dimBlockCarbon.x;
		dim3 dimGridCarbon(numBlocks, 1, 1);
		InitGrapheneCoordsK<<< dimGridCarbon, dimBlockCarbon >>>(dr);
		// copy memory from device to host
		hipMemcpy(hr, dr, hparams->nMol*sizeof(float4), hipMemcpyDeviceToHost);

		// define number of unit cells of metal and number of layers
		if(hparams->nMolMe != 0)
		{
			int initUcellMeX, initUcellMeY, numOfLayers;		
			initUcellMeX = ( hparams->region.x - 2*hparams->a ) / hparams->a;
			initUcellMeY = ( hparams->region.y - 2*hparams->a ) / hparams->a;
			numOfLayers = ceil(hparams->nMolMe*0.25 /(initUcellMeX*initUcellMeY) );

			hparams->initUcellMeX = initUcellMeX;
			hparams->initUcellMeY = initUcellMeY;
			
			dim3 blockMe(1, 1, numOfLayers);
			dim3 gridMe(initUcellMeX, initUcellMeY, 1);

			InitSlabCoordsK<<<gridMe, blockMe>>>( dr );
			// copy memory from device to host
			hipMemcpy(hr, dr, hparams->nMolMe*sizeof(float4), hipMemcpyDeviceToHost);
		}		
	} 	
	hipFree(dr);
	// check errors
	hipError_t error;
	error = hipGetLastError();
	if( error != hipSuccess)
	{
		const char* errorString = hipGetErrorString (error);
		return errorString;
	}
	else return 0;
}

// define color for atoms
//void SetColorW(float4 *color, SimParams* hparams)
//{
	//dim3 dimGrid(hparams->gridSize, 1, 1);		// define number of blocks as in Anderson		
	//dim3 dimBlock(hparams->blockSize, 1, 1);	// number of threads

	//SetColorK <<< dimGrid, dimBlock >>> (color);	
//}

/////////////////////////
// wrappers for CUDA apis
/////////////////////////
void CudaInitW(int argc, char **argv)
{   
//#if __DEVICE_EMULATION__
   //cutilDeviceInit(argc, argv);
//#else
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
   // if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
       // cutilDeviceInit(argc, argv);
    //} else {
        //hipSetDevice( cutGetMaxGflopsDeviceId() );
    //}
	// get cuda device properties
	 int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }
	// Set the first GPU as the working one.
	int gpuId = 0;
	hipSetDevice( gpuId );
	hipGetDeviceProperties(&gDeviceProp, 0);	
//#endif
}

/*
void CudaGLInitW(int argc, char **argv)
{   
#if __DEVICE_EMULATION__
   cutilDeviceInit(argc, argv);
#else
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
    }
#endif
	// get cuda device properties
	hipGetDeviceProperties(&gDeviceProp, 0);
}

void RegisterGLBufferObjectW(uint vbo)
{
    cutilSafeCall(cudaGLRegisterBufferObject(vbo));
}

void UnregisterGLBufferObjectW(uint vbo)
{
    cutilSafeCall(cudaGLUnregisterBufferObject(vbo));
}

void *MapGLBufferObjectW(uint vbo)
{
    void *ptr;
    cutilSafeCall(cudaGLMapBufferObject(&ptr, vbo));
    return ptr;
}

void UnmapGLBufferObjectW(uint vbo)
{
    cutilSafeCall(cudaGLUnmapBufferObject(vbo));
}
*/

void SetParametersW(SimParams *hostParams)
{
	// Allocate device memory.
	//SimParams *deviceParams;
	hipError_t error_id;// = 
		//hipMalloc((void**)&deviceParams,sizeof(SimParams));
	//if (error_id != hipSuccess){
       // printf("Failed to allocate memory %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
       // exit(EXIT_FAILURE);
    //}
	// Copy host data to the device.
	//error_id = hipMemcpy(deviceParams,hostParams,sizeof(SimParams),hipMemcpyHostToDevice);
   // if (error_id != hipSuccess){
      //  printf("Failed to copy memory %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
      //  exit(EXIT_FAILURE);
   // }

    // copy parameters to constant memory
    //cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dparams), hostParams, sizeof(SimParams)) );
	error_id = 
		hipMemcpyToSymbol((const void*)&dparams, hostParams, sizeof(SimParams));
		//hipMemcpyToSymbol(HIP_SYMBOL(pdparams), deviceParams, sizeof(SimParams));
		//hipMemcpyFromSymbolAsync(dparams, HIP_SYMBOL(hostParams), sizeof(SimParams), 0, hipMemcpyHostToDevice, 0);

    if (error_id != hipSuccess)
    {
        printf("Memory error while copying %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        exit(EXIT_FAILURE);
    }

	//hipFree(deviceParams);
	//hipError_t hipMemcpyFromSymbolAsync(
	//hipMemcpyToSymbol(HIP_SYMBOL(dparams), hostParams, sizeof(SimParams), 0, hipMemcpyHostToDevice);
}

}	// end extern "C"

////////////////////////////////////////////
// some host functions called from wrappers
////////////////////////////////////////////

// accumulate properties
void AccumProps (int icode, SimParams *hparams)
{
  if (icode == 0) {
    PropZero (hparams->totEnergy);
	PropZero (hparams->potEnergy);
    PropZero (hparams->kinEnergy);
	PropZero (hparams->oneStep); 
	PropZero (hparams->centerOfMass);
	PropZero (hparams->cmVel);
	PropZero (hparams->frictForce);
  } else if (icode == 1) {
    PropAccum (hparams->totEnergy);
	PropAccum (hparams->potEnergy);
    PropAccum (hparams->kinEnergy);
	PropAccum (hparams->oneStep);
	PropAccum (hparams->centerOfMass);
	PropAccum (hparams->cmVel);
	PropAccum (hparams->frictForce);  
  } else if (icode == 2) {
    PropAvg (hparams->totEnergy, hparams->stepAvg);
	PropAvg (hparams->potEnergy, hparams->stepAvg);
    PropAvg (hparams->kinEnergy, hparams->stepAvg);
	PropAvg (hparams->oneStep, hparams->stepAvg); 
	PropAvg (hparams->centerOfMass, hparams->stepAvg);
	PropAvg (hparams->cmVel, hparams->stepAvg);
	PropAvg (hparams->frictForce, hparams->stepAvg);
  }
}

// print results in a file
void PrintSummary (FILE *fp, SimParams *hparams)
{
	real totEn, totEnRms, potEn, potEnRms, T, TRms;
	
	// compute values in physical units
	// total energy in eV
	totEn = hparams->totEnergy.sum * hparams->enU;
	totEnRms = hparams->totEnergy.sum2 * hparams->enU;
	// potential energy in eV
	potEn = hparams->potEnergy.sum * hparams->enU;
	potEnRms = hparams->potEnergy.sum2 * hparams->enU;
	// temperature in Kelvin
	T = 2.f*hparams->kinEnergy.sum * hparams->temperatureU / 
		( NDIM*hparams->kB/*(1-1/hparams->nMol)*/ );
	TRms = 2.f*hparams->kinEnergy.sum2 * hparams->temperatureU
		/ ( NDIM*hparams->kB/**(1-1/hparams->nMol)*/ );
	
	// print values in the file
	fprintf (fp,
		"%5d\t %7.7f\t %7.7f\t %7.7f\t %7.7f\t %7.7f\t %7.7f\t %7.7f\t %7.7f\t",
		hparams->stepCount, VCSum (hparams->vSum) / hparams->nMol, 
		totEn,	 totEnRms,
		potEn,	 potEnRms,
		T, TRms,
		hparams->oneStep.sum);

	// tribological properties
	real centerOfMass, frictForce, xParticleSize, yParticleSize, zParticleSize, totalShear;
	centerOfMass = hparams->centerOfMass.sum * hparams->lengthU;
	// here is not force unit but 1 nN dimensionless, so divide, not multiply
	frictForce = hparams->frictForce.sum / hparams->forceU;	
	xParticleSize = hparams->particleSize.x * hparams->lengthU;
	yParticleSize = hparams->particleSize.y * hparams->lengthU;
	zParticleSize = hparams->particleSize.z * hparams->lengthU;
	totalShear = hparams->totalShear / hparams->forceU;

	fprintf (fp,
		"%7.7f\t %7.7f\t %7.6f\t %7.7f\t %7.7f\t %7.7f\t %7.7f\n",
		hparams->cmVel.sum, centerOfMass, frictForce, xParticleSize, yParticleSize,
		zParticleSize, totalShear);

	fflush (fp);		// reset buffer
}

// write coordinates to .pdb (protein data bank) file for future use with VMD
int CreatePdbFile(char *szPdb, SimParams *hparams, float4 *r)
{	
	int i;
	int n = 0;
	real m = 0.f;
	TCHAR szFileName[MAX_PATH],szBuf[MAX_PATH];

	ZeroMemory(szBuf, MAX_PATH);

	// to avoid problems with file exstentions
	i = hparams->stepCount / hparams->stepPdb;
	if(i < 100)
		sprintf(szBuf, TEXT("_%i"), i);
	else if(i < 1000)
		sprintf(szBuf, TEXT("_%-3i"), i);
	else if(i < 10000)
		sprintf(szBuf, TEXT("_%-4i"), i);

	lstrcat(szBuf, TEXT(".pdb"));
	lstrcpy(szFileName, szPdb);
	lstrcat(szFileName, szBuf);	

	FILE *pdb = fopen(szFileName, "w");		// Use standard function to open the file	
	if( !pdb )
	{
		//printf("Failed to create output file!");
		return 0;
	}

	// Print information to pdb file accordingly to its format	
	for(i = 0; i < hparams->nMol; i++)
	{   		
										//		Name				   Position in a file
		fprintf(pdb, "ATOM  ");			// Record name colums			1 - 6
		fprintf(pdb, "%-7i", i);		// Atom serial number colums	7 - 11
		if(i >= hparams->nMolMe)
			fprintf(pdb, "C  ");		// Atom name					13 - 16
		else 
			fprintf(pdb, "%2s ", hparams->szNameMe);				
		fprintf(pdb, " ");				// Alternate location indicator 17
		if(i >= hparams->nMolMe)
			fprintf(pdb, " C   ");		// Residue name					18 - 20 ? 21
		else
			fprintf(pdb, " %2s  ", hparams->szNameMe);
		fprintf(pdb, " ");				// Chain identifier				22
		fprintf(pdb, "    ");			// Residue sequence number		23 - 26
		
		fprintf(pdb, "%-4i", n);    	// Code for insertion of residues 27 - 30
		fprintf(pdb, "%-8.2lf", 1.42 *	// Output is in angstoms, so multiply by unit 
					r[i].x);			// Orthogonal coordinates for X  
										// in Angstroms					31 - 38
		fprintf(pdb, "%-8.2lf",	1.42 * 
					r[i].y);			// Orthogonal coordinates for Y  
										// in Angstroms					39 - 46
		// for SG print smaller coordinate for non deposited atoms
		if( (hparams->iRegime == 1) && (i < hparams->nMolMe) && (i >= hparams->nMolDeposited) )
			fprintf(pdb, "%-8.2lf",	1.42 * 2*
			hparams->region.z  );	// Orthogonal coordinates for Z  
										// in Angstroms					47 - 54
		else
			fprintf(pdb, "%-8.2lf",	1.42 * 
					r[i].z);			// Orthogonal coordinates for Z  
										// in Angstroms					47 - 54
		
		fprintf(pdb, " %4.1lf",	m);		// Occupancy					55 - 60		
		fprintf(pdb, "  1.00");		    // Temperature factor			61 - 66
		fprintf(pdb, "          ");		// Not documented				67 - 76
		if(i >= hparams->nMolMe)
			fprintf(pdb, " C");			// Element symbol, right-justified	77 - 78
		else
			fprintf(pdb, "%2s", hparams->szNameMe);

		fprintf(pdb, "  \n");			// Charge on the atom			79 - 80
	}

	fclose(pdb);

	return 1;
}

void PrintRdf(SimParams *hparams, uint *hHistRdf)
{
	real rb;
	int n;
	TCHAR szFileName[MAX_PATH], szBuf[MAX_PATH];
	ZeroMemory(szFileName, MAX_PATH);
	// define filename
	sprintf(szBuf, TEXT("_stepCount_%i_"), hparams->stepCount);	
	lstrcpy(szFileName, hparams->szRdfPath);
	lstrcat(szBuf, hparams->szNameMe);		// add metal name
	lstrcat(szFileName,szBuf);
	lstrcat(szFileName,TEXT(".txt"));
	// this code in Rapaport is outside of PrintRdf
	// but to avoid using real array for histRdf I put this code inside PrintRdf
	real normFac = VProd(hparams->particleSize)*Cube(hparams->intervalRdf) / 
				(2.f * M_PI * hparams->countRdf);
	// to avoid negative values, divide sequentially
	normFac = normFac/hparams->nMolMe;
	normFac = normFac/hparams->nMolMe;

	FILE *rdf = fopen(szFileName,TEXT("w+"));		// Use standard function to open the file

	real histRdf = 0.f;	
	
	for(n = 0; n < hparams->sizeHistRdf; n++)
	{
		rb = (n + 0.5f)*hparams->rangeRdf*hparams->lengthU / hparams->sizeHistRdf;
		histRdf = (real) hHistRdf[n]*normFac / ((n - 0.5f)*(n - 0.5f));
		fprintf(rdf, TEXT("%8.4f\t %8.4f\n"), rb, histRdf);
	}

	fclose(rdf);
}

// generates uniformly distributed random number as VRandR
void VRandRfloat4 (float4 *p, SimParams *hparams)
{
  real s, x, y;

  s = 2.f;
  while (s > 1.f) {
    x = 2.f * RandR (hparams) - 1.f;
    y = 2.f * RandR (hparams) - 1.f;
    s = Sqr (x) + Sqr (y);
  }
  p->z = 1.f - 2.f * s;
  s = 2.f * sqrt (1.f - s);
  p->x = s * x;
  p->y = s * y;
}

// host functions for diffusion
// initialize parameters for diffusion coefficient
void InitDiffusion(TBuf *tBuf, real *rrDiffuseAv, SimParams *hparams)
{
	int nb;
	// assign negative values to count as the initial values
	for(nb = 0; nb < hparams->nBuffDiffuse; nb++)
		tBuf[nb].count = -nb * hparams->nValDiffuse / hparams->nBuffDiffuse;
	ZeroDiffusion(rrDiffuseAv, hparams);
}

// reset parameters
void ZeroDiffusion(real *rrDiffuseAv, SimParams *hparams)
{
	int j;

	hparams->countDiffuseAv = 0;
	for(j = 0; j < hparams->nValDiffuse; j++) rrDiffuseAv[j] = 0.;
}

// print diffusion coefficient in a file
void PrintDiffusion(real *rrDiffuseAv, FILE *file, SimParams *hparams)
{
	real tVal;
	int j;

	fprintf(file, TEXT("diffusion\n"));
	for(j = 0; j < hparams->nValDiffuse; j++)
	{
		tVal = j * hparams->stepDiffuse * hparams->deltaT;
		fprintf(file, TEXT("%8.4f %8.4f\n"), tVal, rrDiffuseAv[j]);
	}
}

// accumulate data for diffusion
void AccumDiffusion(TBuf *tBuf, real *rrDiffuseAv, FILE *file, SimParams *hparams)
{
	real fac;
	int j, nb;

	for(nb = 0; nb < hparams->nBuffDiffuse; nb++)	{
		if(tBuf[nb].count == hparams->nValDiffuse)	{
			for(j = 0; j < hparams->nValDiffuse; j++)
				rrDiffuseAv[j] += tBuf[nb].rrDiffuse[j];
			tBuf[nb].count = 0;
			++ hparams->countDiffuseAv;
			if(hparams->countDiffuseAv == hparams->limitDiffuseAv)
			{
				fac = 1. / (NDIM* 2 * hparams->stepDiffuse * 
					hparams->deltaT * hparams->limitDiffuseAv);
				for(j = 1; j < hparams->nValDiffuse; j++)
					rrDiffuseAv[j] *= fac / j;
				PrintDiffusion(rrDiffuseAv, file, hparams);
				ZeroDiffusion(rrDiffuseAv, hparams);
			}
		}
	}
}

// compute diffusion constant
void EvalDiffusion(TBuf *tBuf, real *rrDiffuseAv, FILE *file, SimParams *hparams,
				   float3 centerOfMass)
{
	float3 dr;
	int nb, ni;

	// loop over all measurement sets
	for(nb = 0; nb < hparams->nBuffDiffuse; nb++) {
		if(tBuf[nb].count == 0){
			tBuf[nb].orgR = centerOfMass;
			tBuf[nb].rTrue = centerOfMass;
		}

		if(tBuf[nb].count >= 0) {
			ni = tBuf[nb].count;
			tBuf[nb].rrDiffuse[ni] = 0.;
			VSub(dr, tBuf[nb].rTrue, centerOfMass);			
			VDiv(dr, dr, hparams->region);
			dr.x = Nint(dr.x);
			dr.y = Nint(dr.y);
			dr.z = Nint(dr.z);
			VMul(dr, dr, hparams->region);
			VAdd(tBuf[nb].rTrue, centerOfMass, dr);			
			VSub(dr, tBuf[nb].rTrue, tBuf[nb].orgR);
			tBuf[nb].rrDiffuse[ni] += VLenSq(dr);
		}// end if(tBuf[nb].count >= 0)
		++ tBuf[nb].count;
	} // end of loop over all sets
	AccumDiffusion(tBuf, rrDiffuseAv, file, hparams);
}